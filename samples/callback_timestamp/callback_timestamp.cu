#include "hip/hip_runtime.h"
/*
 * Copyright 2010-2021 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain timestamps
 * using callbacks for CUDA runtime APIs
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <stdlib.h>

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        const char* errstr;                                                    \
        hipDrvGetErrorString(_status, &errstr);                                    \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, errstr);                     \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define CUPTI_CALL(call)                                                        \
do {                                                                            \
    CUptiResult _status = call;                                                 \
    if (_status != CUPTI_SUCCESS) {                                             \
      const char* errstr;                                                       \
      cuptiGetResultString(_status, &errstr);                                   \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",      \
              __FILE__, __LINE__, #call, errstr);                               \
      exit(EXIT_FAILURE);                                                       \
    }                                                                           \
} while (0)

// Structure to hold data collected by callback
typedef struct RuntimeApiTrace_st {
  const char *functionName;
  uint64_t startTimestamp;
  uint64_t endTimestamp;
  size_t memcpy_bytes;
  enum hipMemcpyKind memcpy_kind;
} RuntimeApiTrace_t;

enum launchOrder{ MEMCPY_H2D1, MEMCPY_H2D2, MEMCPY_D2H, KERNEL, THREAD_SYNC, LAUNCH_LAST};

// Vector addition kernel
__global__ void
VecAdd(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

// Initialize a vector
static void
initVec(int *vec, int n)
{
  for (int i = 0; i < n; i++)
    vec[i] = i;
}

void CUPTIAPI
getTimestampCallback(void *userdata, CUpti_CallbackDomain domain,
                     CUpti_CallbackId cbid, const CUpti_CallbackData *cbInfo)
{
  static int memTransCount = 0;
  uint64_t startTimestamp;
  uint64_t endTimestamp;
  RuntimeApiTrace_t *traceData = (RuntimeApiTrace_t*)userdata;

  // Data is collected only for the following API
  if ((cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020) ||
      (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000) ||
      (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaDeviceSynchronize_v3020) ||
      (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaMemcpy_v3020))  {

    // Set pointer depending on API
    if ((cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020) ||
        (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000))
    {
      traceData = traceData + KERNEL;
    }
    else if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaDeviceSynchronize_v3020)
      traceData = traceData + THREAD_SYNC;
    else if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaMemcpy_v3020)
      traceData = traceData + MEMCPY_H2D1 + memTransCount;

    if (cbInfo->callbackSite == CUPTI_API_ENTER) {
      // for a kernel launch report the kernel name, otherwise use the API
      // function name.
      if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020 ||
          cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000)
      {
        traceData->functionName = cbInfo->symbolName;
      }
      else {
        traceData->functionName = cbInfo->functionName;
      }

      // Store parameters passed to hipMemcpy
      if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaMemcpy_v3020) {
        traceData->memcpy_bytes = ((cudaMemcpy_v3020_params *)(cbInfo->functionParams))->count;
        traceData->memcpy_kind = ((cudaMemcpy_v3020_params *)(cbInfo->functionParams))->kind;
      }

      // Collect timestamp for API start
      CUPTI_CALL(cuptiGetTimestamp(&startTimestamp));

      traceData->startTimestamp = startTimestamp;
    }

    if (cbInfo->callbackSite == CUPTI_API_EXIT) {
      // Collect timestamp for API exit
      CUPTI_CALL(cuptiGetTimestamp(&endTimestamp));

      traceData->endTimestamp = endTimestamp;

      // Advance to the next memory transfer operation
      if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaMemcpy_v3020) {
        memTransCount++;
      }
    }
  }
}

static const char *
memcpyKindStr(enum hipMemcpyKind kind)
{
  switch (kind) {
  case hipMemcpyHostToDevice:
    return "HostToDevice";
  case hipMemcpyDeviceToHost:
    return "DeviceToHost";
  default:
    break;
  }

  return "<unknown>";
}

static void
displayTimestamps(RuntimeApiTrace_t *trace)
{
  // Calculate timestamp of kernel based on timestamp from
  // hipDeviceSynchronize() call
  trace[KERNEL].endTimestamp = trace[THREAD_SYNC].endTimestamp;

  printf("startTimeStamp/Duration reported in nano-seconds\n\n");
  printf("Name\t\tStart Time\t\tDuration\tBytes\tKind\n");
  printf("%s\t%llu\t%llu\t\t%llu\t%s\n", trace[MEMCPY_H2D1].functionName,
         (unsigned long long)trace[MEMCPY_H2D1].startTimestamp,
         (unsigned long long)trace[MEMCPY_H2D1].endTimestamp - trace[MEMCPY_H2D1].startTimestamp,
         (unsigned long long)trace[MEMCPY_H2D1].memcpy_bytes,
         memcpyKindStr(trace[MEMCPY_H2D1].memcpy_kind));
  printf("%s\t%llu\t%llu\t\t%llu\t%s\n", trace[MEMCPY_H2D2].functionName,
         (unsigned long long)trace[MEMCPY_H2D2].startTimestamp,
         (unsigned long long)trace[MEMCPY_H2D2].endTimestamp - trace[MEMCPY_H2D2].startTimestamp,
         (unsigned long long)trace[MEMCPY_H2D2].memcpy_bytes,
         memcpyKindStr(trace[MEMCPY_H2D2].memcpy_kind));
  printf("%s\t%llu\t%llu\t\tNA\tNA\n", trace[KERNEL].functionName,
         (unsigned long long)trace[KERNEL].startTimestamp,
         (unsigned long long)trace[KERNEL].endTimestamp - trace[KERNEL].startTimestamp);
  printf("%s\t%llu\t%llu\t\t%llu\t%s\n", trace[MEMCPY_D2H].functionName,
         (unsigned long long)trace[MEMCPY_D2H].startTimestamp,
         (unsigned long long)trace[MEMCPY_D2H].endTimestamp - trace[MEMCPY_D2H].startTimestamp,
         (unsigned long long)trace[MEMCPY_D2H].memcpy_bytes,
         memcpyKindStr(trace[MEMCPY_D2H].memcpy_kind));
}

static void
cleanUp(int *h_A, int *h_B, int *h_C, int *d_A, int *d_B, int *d_C)
{
  if (d_A)
    RUNTIME_API_CALL(hipFree(d_A));
  if (d_B)
    RUNTIME_API_CALL(hipFree(d_B));
  if (d_C)
    RUNTIME_API_CALL(hipFree(d_C));

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_B)
    free(h_B);
  if (h_C)
    free(h_C);
}

int
main()
{
  hipCtx_t context = 0;
  hipDevice_t device = 0;
  int N = 50000;
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int sum, i;
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;

  CUpti_SubscriberHandle subscriber;
  RuntimeApiTrace_t trace[LAUNCH_LAST];

  // subscribe to CUPTI callbacks
  CUPTI_CALL(cuptiSubscribe(&subscriber, (CUpti_CallbackFunc)getTimestampCallback , &trace));

  DRIVER_API_CALL(hipInit(0));

  DRIVER_API_CALL(hipCtxCreate(&context, 0, device));

  // Enable all callbacks for CUDA Runtime APIs.
  // Callback will be invoked at the entry and exit points of each of the CUDA Runtime API
  CUPTI_CALL(cuptiEnableDomain(1, subscriber, CUPTI_CB_DOMAIN_RUNTIME_API));

  // Allocate input vectors h_A and h_B in host memory
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);
  if (!h_A || !h_B || !h_C) {
    printf("Error: out of memory\n");
    exit(EXIT_FAILURE);
  }

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);

  // Allocate vectors in device memory
  RUNTIME_API_CALL(hipMalloc((void**)&d_A, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_B, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_C, size));

  // Copy vectors from host memory to device memory
  RUNTIME_API_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
  RUNTIME_API_CALL(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

  // Invoke kernel
  threadsPerBlock = 256;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
  RUNTIME_API_CALL(hipDeviceSynchronize());

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  RUNTIME_API_CALL(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

  // Verify result
  for (i = 0; i < N; ++i) {
    sum = h_A[i] + h_B[i];
    if (h_C[i] != sum) {
      printf("kernel execution FAILED\n");
      goto Error;
    }
  }

  // display timestamps collected in the callback
  displayTimestamps(trace);

  CUPTI_CALL(cuptiUnsubscribe(subscriber));

  cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
  RUNTIME_API_CALL(hipDeviceSynchronize());
  exit(EXIT_SUCCESS);

 Error:
  cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
  RUNTIME_API_CALL(hipDeviceSynchronize());
  exit(EXIT_FAILURE);
}

