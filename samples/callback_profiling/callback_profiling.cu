#include "hip/hip_runtime.h"
//
// Copyright 2020 NVIDIA Corporation. All rights reserved
//

#include <cupti_target.h>
#include <cupti_profiler_target.h>
#include <cupti_callbacks.h>
#include <cupti_driver_cbid.h>
#include <nvperf_host.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <Metric.h>
#include <Eval.h>
#include <FileOp.h>

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

#define NVPW_API_CALL(apiFuncCall)                                             \
do {                                                                           \
    NVPA_Status _status = apiFuncCall;                                         \
    if (_status != NVPA_STATUS_SUCCESS) {                                      \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define CUPTI_API_CALL(apiFuncCall)                                            \
do {                                                                           \
    CUptiResult _status = apiFuncCall;                                         \
    if (_status != CUPTI_SUCCESS) {                                            \
        const char *errstr;                                                    \
        cuptiGetResultString(_status, &errstr);                                \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, errstr);                     \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                     \
    }                                                                          \
} while (0)

#define METRIC_NAME "sm__ctas_launched.sum"

struct ProfilingData_t
{
    int numRanges = 2;
    bool bProfiling = false;
    std::string chipName;
    std::vector<std::string> metricNames;
    std::string CounterDataFileName = "SimpleCupti.counterdata";
    std::string CounterDataSBFileName = "SimpleCupti.counterdataSB";
    CUpti_ProfilerRange profilerRange = CUPTI_AutoRange;
    CUpti_ProfilerReplayMode profilerReplayMode = CUPTI_UserReplay;
    bool allPassesSubmitted = true;
    std::vector<uint8_t> counterDataImagePrefix;
    std::vector<uint8_t> configImage;
    std::vector<uint8_t> counterDataImage;
    std::vector<uint8_t> counterDataScratchBuffer;
};

__global__
void VecAdd(const int* A, const int* B, int* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void enableProfiling(ProfilingData_t* pProfilingData)
{
    CUpti_Profiler_EnableProfiling_Params enableProfilingParams = { CUpti_Profiler_EnableProfiling_Params_STRUCT_SIZE };
    if (pProfilingData->profilerReplayMode == CUPTI_KernelReplay)
    {
        CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
    }
    else if (pProfilingData->profilerReplayMode == CUPTI_UserReplay)
    {
        CUpti_Profiler_BeginPass_Params beginPassParams = { CUpti_Profiler_BeginPass_Params_STRUCT_SIZE };
        CUPTI_API_CALL(cuptiProfilerBeginPass(&beginPassParams));
        CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
    }
}

void disableProfiling(ProfilingData_t* pProfilingData)
{
    CUpti_Profiler_DisableProfiling_Params disableProfilingParams = { CUpti_Profiler_DisableProfiling_Params_STRUCT_SIZE };
    CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));

    if (pProfilingData->profilerReplayMode == CUPTI_UserReplay)
    {
        CUpti_Profiler_EndPass_Params endPassParams = { CUpti_Profiler_EndPass_Params_STRUCT_SIZE };
        CUPTI_API_CALL(cuptiProfilerEndPass(&endPassParams));
        pProfilingData->allPassesSubmitted = (endPassParams.allPassesSubmitted == 1) ? true : false;
    }
    else if (pProfilingData->profilerReplayMode == CUPTI_KernelReplay)
    {
        pProfilingData->allPassesSubmitted = true;
    }

    if (pProfilingData->allPassesSubmitted)
    {
        CUpti_Profiler_FlushCounterData_Params flushCounterDataParams = { CUpti_Profiler_FlushCounterData_Params_STRUCT_SIZE };
        CUPTI_API_CALL(cuptiProfilerFlushCounterData(&flushCounterDataParams));
    }
}

void beginSession(ProfilingData_t* pProfilingData)
{
    CUpti_Profiler_BeginSession_Params beginSessionParams = { CUpti_Profiler_BeginSession_Params_STRUCT_SIZE };
    beginSessionParams.ctx = NULL;
    beginSessionParams.counterDataImageSize = pProfilingData->counterDataImage.size();
    beginSessionParams.pCounterDataImage = &pProfilingData->counterDataImage[0];
    beginSessionParams.counterDataScratchBufferSize = pProfilingData->counterDataScratchBuffer.size();
    beginSessionParams.pCounterDataScratchBuffer = &pProfilingData->counterDataScratchBuffer[0];
    beginSessionParams.range = pProfilingData->profilerRange;
    beginSessionParams.replayMode = pProfilingData->profilerReplayMode;
    beginSessionParams.maxRangesPerPass = pProfilingData->numRanges;
    beginSessionParams.maxLaunchesPerPass = pProfilingData->numRanges;
    CUPTI_API_CALL(cuptiProfilerBeginSession(&beginSessionParams));
}

void setConfig(ProfilingData_t* pProfilingData)
{
    CUpti_Profiler_SetConfig_Params setConfigParams = { CUpti_Profiler_SetConfig_Params_STRUCT_SIZE };
    setConfigParams.pConfig = &pProfilingData->configImage[0];
    setConfigParams.configSize = pProfilingData->configImage.size();
    setConfigParams.passIndex = 0;
    CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));
}

void createCounterDataImage(int numRanges,
    std::vector<uint8_t>& counterDataImagePrefix,
    std::vector<uint8_t>& counterDataScratchBuffer,
    std::vector<uint8_t>& counterDataImage
)
{
    CUpti_Profiler_CounterDataImageOptions counterDataImageOptions;
    counterDataImageOptions.pCounterDataPrefix = &counterDataImagePrefix[0];
    counterDataImageOptions.counterDataPrefixSize = counterDataImagePrefix.size();
    counterDataImageOptions.maxNumRanges = numRanges;
    counterDataImageOptions.maxNumRangeTreeNodes = numRanges;
    counterDataImageOptions.maxRangeNameLength = 64;

    CUpti_Profiler_CounterDataImage_CalculateSize_Params calculateSizeParams = { CUpti_Profiler_CounterDataImage_CalculateSize_Params_STRUCT_SIZE };
    calculateSizeParams.pOptions = &counterDataImageOptions;
    calculateSizeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateSize(&calculateSizeParams));

    CUpti_Profiler_CounterDataImage_Initialize_Params initializeParams = { CUpti_Profiler_CounterDataImage_Initialize_Params_STRUCT_SIZE };
    initializeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    initializeParams.pOptions = &counterDataImageOptions;
    initializeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    counterDataImage.resize(calculateSizeParams.counterDataImageSize);
    initializeParams.pCounterDataImage = &counterDataImage[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitialize(&initializeParams));

    CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params scratchBufferSizeParams = { CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params_STRUCT_SIZE };
    scratchBufferSizeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    scratchBufferSizeParams.pCounterDataImage = initializeParams.pCounterDataImage;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateScratchBufferSize(&scratchBufferSizeParams));
    counterDataScratchBuffer.resize(scratchBufferSizeParams.counterDataScratchBufferSize);

    CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params initScratchBufferParams = { CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params_STRUCT_SIZE };
    initScratchBufferParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    initScratchBufferParams.pCounterDataImage = initializeParams.pCounterDataImage;
    initScratchBufferParams.counterDataScratchBufferSize = scratchBufferSizeParams.counterDataScratchBufferSize;
    initScratchBufferParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitializeScratchBuffer(&initScratchBufferParams));
}

void setupProfiling(ProfilingData_t* pProfilingData)
{
    /* Generate configuration for metrics, this can also be done offline*/
    NVPW_InitializeHost_Params initializeHostParams = { NVPW_InitializeHost_Params_STRUCT_SIZE };
    NVPW_API_CALL(NVPW_InitializeHost(&initializeHostParams));

    if (pProfilingData->metricNames.size())
    {
        if (!NV::Metric::Config::GetConfigImage(pProfilingData->chipName, pProfilingData->metricNames, pProfilingData->configImage))
        {
            std::cout << "Failed to create configImage" << std::endl;
            exit(EXIT_FAILURE);
        }
        if (!NV::Metric::Config::GetCounterDataPrefixImage(pProfilingData->chipName, pProfilingData->metricNames, pProfilingData->counterDataImagePrefix))
        {
            std::cout << "Failed to create counterDataImagePrefix" << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    else
    {
        std::cout << "No metrics provided to profile" << std::endl;
        exit(EXIT_FAILURE);
    }

    createCounterDataImage(pProfilingData->numRanges, pProfilingData->counterDataImagePrefix,
                           pProfilingData->counterDataScratchBuffer, pProfilingData->counterDataImage);

    beginSession(pProfilingData);
    setConfig(pProfilingData);
}

void stopProfiling(ProfilingData_t* pProfilingData)
{
    CUpti_Profiler_UnsetConfig_Params unsetConfigParams = { CUpti_Profiler_UnsetConfig_Params_STRUCT_SIZE };
    CUpti_Profiler_EndSession_Params endSessionParams = { CUpti_Profiler_EndSession_Params_STRUCT_SIZE };
    CUpti_Profiler_DeInitialize_Params profilerDeInitializeParams = {CUpti_Profiler_DeInitialize_Params_STRUCT_SIZE};

    CUPTI_API_CALL(cuptiProfilerUnsetConfig(&unsetConfigParams));
    CUPTI_API_CALL(cuptiProfilerEndSession(&endSessionParams));
    CUPTI_API_CALL(cuptiProfilerDeInitialize(&profilerDeInitializeParams));

    // Dump counterDataImage and counterDataScratchBuffer in file.
    WriteBinaryFile(pProfilingData->CounterDataFileName.c_str(), pProfilingData->counterDataImage);
    WriteBinaryFile(pProfilingData->CounterDataSBFileName.c_str(), pProfilingData->counterDataScratchBuffer);
}

void callbackHandler(void* userdata, CUpti_CallbackDomain domain,
                      CUpti_CallbackId cbid, void* cbdata)
{
    ProfilingData_t* profilingData = (ProfilingData_t*)(userdata);
    const CUpti_CallbackData* cbInfo = (CUpti_CallbackData*)cbdata;
    switch (domain)
    {
    case CUPTI_CB_DOMAIN_DRIVER_API:
        switch (cbid)
        {
        case CUPTI_DRIVER_TRACE_CBID_cuLaunchKernel:
        {
            if (cbInfo->callbackSite == CUPTI_API_ENTER)
            {
                enableProfiling(profilingData);
            }
            else
            {
                disableProfiling(profilingData);
            }
        }
        break;
        default:
            break;
        }
        break;
    case CUPTI_CB_DOMAIN_RESOURCE:
        switch (cbid)
        {
        case CUPTI_CBID_RESOURCE_CONTEXT_CREATED:
        {
            setupProfiling(profilingData);
            profilingData->bProfiling = true;
        }
        break;
        default:
            break;
        }
        break;
    default:
        break;
    }

}

void initVec(int* vec, int n)
{
    for (int i = 0; i < n; i++)
        vec[i] = i;
}

void cleanUp(int* h_A, int* h_B, int* h_C, int* d_A, int* d_B, int* d_C)
{
    if (d_A)
        RUNTIME_API_CALL(hipFree(d_A));
    if (d_B)
        RUNTIME_API_CALL(hipFree(d_B));
    if (d_C)
        RUNTIME_API_CALL(hipFree(d_C));

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
}

void VectorAdd()
{
    int N = 50000;
    size_t size = N * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int* h_A, * h_B, * h_C;
    int* d_A, * d_B, * d_C;
    int i, sum;

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize input vectors
    initVec(h_A, N);
    initVec(h_B, N);
    memset(h_C, 0, size);

    // Allocate vectors in device memory
    RUNTIME_API_CALL(hipMalloc((void**)&d_A, size));
    RUNTIME_API_CALL(hipMalloc((void**)&d_B, size));
    RUNTIME_API_CALL(hipMalloc((void**)&d_C, size));

    // Copy vectors from host memory to device memory
    RUNTIME_API_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    RUNTIME_API_CALL(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // Invoke kernel
    threadsPerBlock = 256;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("Launching kernel: blocks %d, thread/block %d\n",
        blocksPerGrid, threadsPerBlock);

    VecAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    RUNTIME_API_CALL(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // Verify result
    for (i = 0; i < N; ++i)
    {
        sum = h_A[i] + h_B[i];
        if (h_C[i] != sum)
        {
            fprintf(stderr, "error: result verification failed\n");
            exit(EXIT_FAILURE);
        }
    }

    cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
}

int main(int argc, char* argv[])
{
    hipDevice_t cuDevice = 0;
    int deviceCount, deviceNum = 0;
    int computeCapabilityMajor = 0, computeCapabilityMinor = 0;

    DRIVER_API_CALL(hipInit(0));
    DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0)
    {
        printf("There is no device supporting CUDA.\n");
        exit(EXIT_WAIVED);
    }

    DRIVER_API_CALL(hipDeviceGet(&cuDevice, deviceNum));

    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, cuDevice));
    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, cuDevice));
    printf("Compute Capability of Device: %d.%d\n", computeCapabilityMajor, computeCapabilityMinor);

    // Initialize profiler API support and test device compatibility
    CUpti_Profiler_Initialize_Params profilerInitializeParams = {CUpti_Profiler_Initialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerInitialize(&profilerInitializeParams));
    CUpti_Profiler_DeviceSupported_Params params = { CUpti_Profiler_DeviceSupported_Params_STRUCT_SIZE };
    params.cuDevice = deviceNum;
    CUPTI_API_CALL(cuptiProfilerDeviceSupported(&params));

    if (params.isSupported != CUPTI_PROFILER_CONFIGURATION_SUPPORTED)
    {
        ::std::cerr << "Unable to profile on device " << deviceNum << ::std::endl;

        if (params.architecture == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice architecture is not supported" << ::std::endl;
        }

        if (params.sli == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice sli configuration is not supported" << ::std::endl;
        }

        if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice vgpu configuration is not supported" << ::std::endl;
        }
        else if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_DISABLED)
        {
            ::std::cerr << "\tdevice vgpu configuration disabled profiling support" << ::std::endl;
        }

        if (params.confidentialCompute == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice confidential compute configuration is not supported" << ::std::endl;
        }

        if (params.cmp == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tNVIDIA Crypto Mining Processors (CMP) are not supported" << ::std::endl;
        }
        exit(EXIT_WAIVED);
    }

    ProfilingData_t* profilingData = new ProfilingData_t();
    for (int i = 1; i < argc; ++i)
    {
        char* arg = argv[i];
        if (strcmp(arg, "--help") == 0 || strcmp(arg, "-h") == 0)
        {
            printf("Usage: %s -d [device_num] -m [metric_names comma separated] -n [num of ranges] -r [kernel or user] -o [counterdata filename]\n", argv[0]);
            exit(EXIT_SUCCESS);
        }

        if (strcmp(arg, "--device") == 0 || strcmp(arg, "-d") == 0)
        {
            deviceNum = atoi(argv[i + 1]);
            printf("CUDA Device Number: %d\n", deviceNum);
            i++;
        }
        else if (strcmp(arg, "--metrics") == 0 || strcmp(arg, "-m") == 0)
        {
            char* metricName = strtok(argv[i + 1], ",");
            while (metricName != NULL)
            {
                profilingData->metricNames.push_back(metricName);
                metricName = strtok(NULL, ",");
            }
            i++;
        }
        else if (strcmp(arg, "--numRanges") == 0 || strcmp(arg, "-n") == 0)
        {
            int numRanges = atoi(argv[i + 1]);
            profilingData->numRanges = numRanges;
            i++;
        }
        else if (strcmp(arg, "--replayMode") == 0 || strcmp(arg, "-r") == 0)
        {
            std::string replayMode(argv[i + 1]);
            if (replayMode == "kernel")
                profilingData->profilerReplayMode = CUPTI_KernelReplay;
            else if (replayMode == "user")
                profilingData->profilerReplayMode = CUPTI_UserReplay;
            else {
                printf("Invalid --replayMode argument supported replayMode type 'kernel' or 'user'\n");
                exit(EXIT_FAILURE);
            }
            i++;
        }
        else if (strcmp(arg, "--outputCounterData") == 0 || strcmp(arg, "-o") == 0)
        {
            std::string outputCounterData(argv[i + 1]);
            profilingData->CounterDataFileName = outputCounterData;
            profilingData->CounterDataSBFileName = outputCounterData + "SB";
            i++;
        }
        else {
            printf("Error!! Invalid Arguments\n");
            printf("Usage: %s -d [device_num] -m [metric_names comma separated] -n [num of ranges] -r [kernel or user] -o [counterdata filename]\n", argv[0]);
            exit(EXIT_FAILURE);
        }
    }

    if (profilingData->metricNames.empty())
        profilingData->metricNames.push_back(METRIC_NAME);

    CUpti_Device_GetChipName_Params getChipNameParams = { CUpti_Device_GetChipName_Params_STRUCT_SIZE };
    getChipNameParams.deviceIndex = deviceNum;
    CUPTI_API_CALL(cuptiDeviceGetChipName(&getChipNameParams));
    profilingData->chipName = getChipNameParams.pChipName;

    CUpti_SubscriberHandle subscriber;
    CUPTI_API_CALL(cuptiSubscribe(&subscriber, (CUpti_CallbackFunc)callbackHandler, profilingData));
    CUPTI_API_CALL(cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_RESOURCE, CUPTI_CBID_RESOURCE_CONTEXT_CREATED));
    CUPTI_API_CALL(cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_DRIVER_API, CUPTI_DRIVER_TRACE_CBID_cuLaunchKernel));

    do
    {
        VectorAdd();
    }
    while (!profilingData->allPassesSubmitted);

    if (profilingData->bProfiling)
    {
        stopProfiling(profilingData);
        profilingData->bProfiling = false;

        /* Evaluation of metrics collected in counterDataImage, this can also be done offline*/
        NV::Metric::Eval::PrintMetricValues(profilingData->chipName, profilingData->counterDataImage, profilingData->metricNames);
    }

    delete profilingData;
    exit(EXIT_SUCCESS);
}
