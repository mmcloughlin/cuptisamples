/*
 * Copyright 2021 NVIDIA Corporation. All rights reserved
 *
 * Sample CUPTI app to print trace of CUDA memory operations.
 * The sample also traces CUDA memory operations done via
 * default memory pool.
 *
 */

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

#define DRIVER_API_CALL(apiFuncCall)                                                \
    do {                                                                            \
        hipError_t _status = apiFuncCall;                                             \
        if (_status != hipSuccess) {                                              \
            const char* errstr;                                                     \
            hipDrvGetErrorString(_status, &errstr);                                     \
            fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",    \
                    __FILE__, __LINE__, #apiFuncCall, errstr);                      \
            exit(EXIT_FAILURE);                                                      \
        }                                                                           \
    } while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                               \
    do {                                                                            \
        hipError_t _status = apiFuncCall;                                          \
        if (_status != hipSuccess) {                                               \
            fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",    \
                    __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status)); \
            exit(EXIT_FAILURE);                                                     \
        }                                                                           \
    } while (0)

extern void initTrace(void);
extern void finiTrace(void);

__global__ void vectorAddGPU(const float *a, const float *b, float *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

static void
memoryAllocations()
{
    int nelem = 1048576;
    size_t size = nelem * sizeof(int);

    int *h_A, *h_B;
    int *d_A, *d_B;

    // Allocate memory
    RUNTIME_API_CALL(hipHostMalloc((void**)&h_A, size, hipHostMallocDefault));
    RUNTIME_API_CALL(hipHostAlloc((void**)&h_B, size, hipHostMallocPortable));
    RUNTIME_API_CALL(hipMalloc((void**)&d_A, size));
    RUNTIME_API_CALL(hipMallocManaged((void**)&d_B, size, hipMemAttachGlobal));

    // Free the allocated memory
    RUNTIME_API_CALL(hipHostFree(h_A));
    RUNTIME_API_CALL(hipHostFree(h_B));
    RUNTIME_API_CALL(hipFree(d_A));
    RUNTIME_API_CALL(hipFree(d_B));
}

static void
memoryAllocationsViaMemoryPool()
{
    int nelem = 1048576;
    size_t bytes = nelem * sizeof(float);

    float *a, *b, *c;
    float *d_A, *d_B, *d_C;
    hipStream_t stream;

    int isMemPoolSupported = 0;
    hipError_t status = hipSuccess;
    status = hipDeviceGetAttribute(&isMemPoolSupported, hipDeviceAttributeMemoryPoolsSupported, 0);
    // For enhance compatibility cases, the attribute cudaDevAttrMemoryPoolsSupported might not be present
    // return early if Runtime API does not return cudaSuccess
    if (!isMemPoolSupported || status != hipSuccess) {
        printf("Warning: Waiving execution of memory operations via memory pool as device does not support memory pools.\n");
        return;
    }

    // Allocate and initialize memory on host and device
    a = (float*) malloc(bytes);
    b = (float*) malloc(bytes);
    c = (float*) malloc(bytes);

    for (int n = 0; n < nelem; n++) {
        a[n] = rand() / (float)RAND_MAX;
        b[n] = rand() / (float)RAND_MAX;
    }

    RUNTIME_API_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // Allocate memory using default memory pool
    RUNTIME_API_CALL(hipMallocAsync(&d_A, bytes, stream));
    RUNTIME_API_CALL(hipMallocAsync(&d_B, bytes, stream));
    RUNTIME_API_CALL(hipMallocAsync(&d_C, bytes, stream));
    RUNTIME_API_CALL(hipMemcpyAsync(d_A, a, bytes, hipMemcpyHostToDevice, stream));
    RUNTIME_API_CALL(hipMemcpyAsync(d_B, b, bytes, hipMemcpyHostToDevice, stream));

    dim3 block(256);
    dim3 grid((unsigned int)ceil(nelem/(float)block.x));
    vectorAddGPU <<< grid, block, 0, stream >>>(d_A, d_B, d_C, nelem);

    // Free the allocated memory
    RUNTIME_API_CALL(hipFreeAsync(d_A, stream));
    RUNTIME_API_CALL(hipFreeAsync(d_B, stream));
    RUNTIME_API_CALL(hipMemcpyAsync(c, d_C, bytes, hipMemcpyDeviceToHost, stream));
    RUNTIME_API_CALL(hipFree(d_C));

    RUNTIME_API_CALL(hipStreamSynchronize(stream));
    RUNTIME_API_CALL(hipStreamDestroy(stream));

    free(a);
    free(b);
    free(c);
}

int
main(int argc, char *argv[])
{
    // Initialize CUPTI
    initTrace();

    // Initialize CUDA
    DRIVER_API_CALL(hipInit(0));

    char deviceName[256];
    hipDevice_t device;
    DRIVER_API_CALL(hipDeviceGet(&device, 0));
    DRIVER_API_CALL(hipDeviceGetName(deviceName, 256, device));
    printf("Device Name: %s\n", deviceName);
    RUNTIME_API_CALL(hipSetDevice(0));

    memoryAllocations();
    memoryAllocationsViaMemoryPool();

    // Flush CUPTI activity buffers
    finiTrace();

    exit(EXIT_SUCCESS);
}

