#include "hip/hip_runtime.h"
/*
 * Copyright 2010-2017 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain profiler event values
 * using callbacks for CUDA runtime APIs
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <stdlib.h>

#define EVENT_NAME "inst_executed"

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

#define CHECK_CU_ERROR(err, cufunc)                                     \
  if (err != hipSuccess)                                              \
    {                                                                   \
      printf ("%s:%d: error %d for CUDA Driver API function '%s'\n",    \
              __FILE__, __LINE__, err, cufunc);                         \
      exit(EXIT_FAILURE);                                                \
    }

#define CHECK_CUPTI_ERROR(err, cuptifunc)                               \
  if (err != CUPTI_SUCCESS)                                             \
    {                                                                   \
      const char *errstr;                                               \
      cuptiGetResultString(err, &errstr);                               \
      printf ("%s:%d:Error %s for CUPTI API function '%s'.\n",          \
              __FILE__, __LINE__, errstr, cuptifunc);                   \
      exit(EXIT_FAILURE);                                               \
    }

typedef struct cupti_eventData_st {
  CUpti_EventGroup eventGroup;
  CUpti_EventID eventId;
} cupti_eventData;

// Structure to hold data collected by callback
typedef struct RuntimeApiTrace_st {
  cupti_eventData *eventData;
  uint64_t eventVal;
} RuntimeApiTrace_t;

// Device code
__global__ void VecAdd(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

static void
initVec(int *vec, int n)
{
  for (int i=0; i< n; i++)
    vec[i] = i;
}

void CUPTIAPI
getEventValueCallback(void *userdata, CUpti_CallbackDomain domain,
                      CUpti_CallbackId cbid, const CUpti_CallbackData *cbInfo)
{
  CUptiResult cuptiErr;
  RuntimeApiTrace_t *traceData = (RuntimeApiTrace_t*)userdata;
  size_t bytesRead;

  // This callback is enabled only for launch so we shouldn't see anything else.
  if ((cbid != CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020) &&
      (cbid != CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000))
  {
    printf("%s:%d: unexpected cbid %d\n", __FILE__, __LINE__, cbid);
    exit(EXIT_FAILURE);
  }

  if (cbInfo->callbackSite == CUPTI_API_ENTER) {
    hipDeviceSynchronize();
    cuptiErr = cuptiSetEventCollectionMode(cbInfo->context,
                                           CUPTI_EVENT_COLLECTION_MODE_KERNEL);
    CHECK_CUPTI_ERROR(cuptiErr, "cuptiSetEventCollectionMode");
    cuptiErr = cuptiEventGroupEnable(traceData->eventData->eventGroup);
    CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupEnable");
  }

  if (cbInfo->callbackSite == CUPTI_API_EXIT) {
    uint32_t numInstances = 0, i;
    uint64_t *values = NULL;
    size_t valueSize = sizeof(numInstances);

    cuptiErr = cuptiEventGroupGetAttribute(traceData->eventData->eventGroup,
                                           CUPTI_EVENT_GROUP_ATTR_INSTANCE_COUNT,
                                           &valueSize, &numInstances);
    CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupGetAttribute");

    bytesRead = sizeof (uint64_t) * numInstances;
    values = (uint64_t *) malloc(bytesRead);
    if (values == NULL) {
        printf("%s:%d: Out of memory\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
    cuptiErr = cuptiEventGroupReadEvent(traceData->eventData->eventGroup,
                                        CUPTI_EVENT_READ_FLAG_NONE,
                                        traceData->eventData->eventId,
                                        &bytesRead, values);
    CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupReadEvent");

    traceData->eventVal = 0;
    for (i=0; i<numInstances; i++) {
        traceData->eventVal += values[i];
    }
    free(values);

    cuptiErr = cuptiEventGroupDisable(traceData->eventData->eventGroup);
    CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDisable");
  }
}

static void
displayEventVal(RuntimeApiTrace_t *trace, const char *eventName)
{
  printf("Event Name : %s \n", eventName);
  printf("Event Value : %llu\n", (unsigned long long) trace->eventVal);
}

static void
cleanUp(int *h_A, int *h_B, int *h_C, int *d_A, int *d_B, int *d_C)
{
  if (d_A)
    hipFree(d_A);
  if (d_B)
    hipFree(d_B);
  if (d_C)
    hipFree(d_C);

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_B)
    free(h_B);
  if (h_C)
    free(h_C);
}

int
main(int argc, char *argv[])
{
  hipCtx_t context = 0;
  hipDevice_t dev = 0;
  hipError_t err;
  int N = 50000;
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int sum, i;
  int computeCapabilityMajor=0;
  int computeCapabilityMinor=0;
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;
  int deviceNum;
  int deviceCount;
  char deviceName[256];
  const char *eventName;
  uint32_t profile_all = 1;

  CUptiResult cuptiErr;
  CUpti_SubscriberHandle subscriber;
  cupti_eventData cuptiEvent;
  RuntimeApiTrace_t trace;

  printf("Usage: %s [device_num] [event_name]\n", argv[0]);

  err = hipInit(0);
  CHECK_CU_ERROR(err, "hipInit");

  err = hipGetDeviceCount(&deviceCount);
  CHECK_CU_ERROR(err, "hipGetDeviceCount");

  if (deviceCount == 0) {
    printf("There is no device supporting CUDA.\n");
    exit(EXIT_WAIVED);
  }

  if (argc > 1)
    deviceNum = atoi(argv[1]);
  else
    deviceNum = 0;
  printf("CUDA Device Number: %d\n", deviceNum);

  err = hipDeviceGet(&dev, deviceNum);
  CHECK_CU_ERROR(err, "hipDeviceGet");

  err = hipDeviceGetName(deviceName, 256, dev);
  CHECK_CU_ERROR(err, "hipDeviceGetName");

  printf("CUDA Device Name: %s\n", deviceName);

  err = hipDeviceGetAttribute(&computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, dev);
  CHECK_CU_ERROR(err, "hipDeviceGetAttribute");

  err = hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, dev);
  CHECK_CU_ERROR(err, "hipDeviceGetAttribute");

  printf("Compute Capability of Device: %d.%d\n", computeCapabilityMajor, computeCapabilityMinor);
  int deviceComputeCapability = 10 * computeCapabilityMajor + computeCapabilityMinor;
  if(deviceComputeCapability > 72) {
    printf("Sample unsupported on Device with compute capability > 7.2\n");
    exit(EXIT_WAIVED);
  }

  err = hipCtxCreate(&context, 0, dev);
  CHECK_CU_ERROR(err, "hipCtxCreate");


  // Creating event group for profiling
  cuptiErr = cuptiEventGroupCreate(context, &cuptiEvent.eventGroup, 0);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupCreate");

  if (argc > 2) {
      eventName = argv[2];
  }
  else {
      eventName = EVENT_NAME;
  }

  cuptiErr = cuptiEventGetIdFromName(dev, eventName, &cuptiEvent.eventId);
  if (cuptiErr != CUPTI_SUCCESS)
    {
      printf("Invalid eventName: %s\n", eventName);
      exit(EXIT_FAILURE);
    }

  cuptiErr = cuptiEventGroupAddEvent(cuptiEvent.eventGroup, cuptiEvent.eventId);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupAddEvent");

  cuptiErr = cuptiEventGroupSetAttribute(cuptiEvent.eventGroup,
                                         CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
                                         sizeof(profile_all), &profile_all);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupSetAttribute");

  trace.eventData = &cuptiEvent;

  cuptiErr = cuptiSubscribe(&subscriber, (CUpti_CallbackFunc)getEventValueCallback , &trace);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiSubscribe");

  cuptiErr = cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_RUNTIME_API,
                                 CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEnableCallback");
  cuptiErr = cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_RUNTIME_API,
                                 CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEnableCallback");

  // Allocate input vectors h_A and h_B in host memory
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel
  threadsPerBlock = 256;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  printf("Launching kernel: blocks %d, thread/block %d\n",
         blocksPerGrid, threadsPerBlock);

  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // Verify result
  for (i = 0; i < N; ++i) {
    sum = h_A[i] + h_B[i];
    if (h_C[i] != sum) {
      printf("kernel execution FAILED\n");
      goto Error;
    }
  }

  displayEventVal(&trace, eventName);

  trace.eventData = NULL;

  cuptiErr = cuptiEventGroupRemoveEvent(cuptiEvent.eventGroup, cuptiEvent.eventId);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupRemoveEvent");

  cuptiErr = cuptiEventGroupDestroy(cuptiEvent.eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDestroy");

  cuptiErr = cuptiUnsubscribe(subscriber);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiUnsubscribe");

  cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
  hipDeviceSynchronize();
  exit(EXIT_SUCCESS);

 Error:
  cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
  hipDeviceSynchronize();
  exit(EXIT_FAILURE);
}

