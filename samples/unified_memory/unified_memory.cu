/*
 * Copyright 2014-2015 NVIDIA Corporation. All rights reserved.
 *
 * Sample CUPTI app to demonstrate the usage of unified memory counter profiling
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cupti.h>

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

#define CUPTI_CALL(call)                                                    \
do {                                                                        \
    CUptiResult _status = call;                                             \
    if (_status != CUPTI_SUCCESS) {                                         \
      const char *errstr;                                                   \
      cuptiGetResultString(_status, &errstr);                               \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",  \
              __FILE__, __LINE__, #call, errstr);                           \
      exit(EXIT_FAILURE);                                                    \
    }                                                                       \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                     \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define BUF_SIZE (8 * 1024)
#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \
    (((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer))

static const char *
getUvmCounterKindString(CUpti_ActivityUnifiedMemoryCounterKind kind)
{
    switch (kind)
    {
    case CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_HTOD:
        return "BYTES_TRANSFER_HTOD";
    case CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_DTOH:
        return "BYTES_TRANSFER_DTOH";
    default:
        break;
    }
    return "<unknown>";
}

static void
printActivity(CUpti_Activity *record)
{
    switch (record->kind)
    {
    case CUPTI_ACTIVITY_KIND_UNIFIED_MEMORY_COUNTER:
        {
            CUpti_ActivityUnifiedMemoryCounter2 *uvm = (CUpti_ActivityUnifiedMemoryCounter2 *)record;
            printf("UNIFIED_MEMORY_COUNTER [ %llu %llu ] kind=%s value=%llu src %u dst %u\n",
                (unsigned long long)(uvm->start),
                (unsigned long long)(uvm->end),
                getUvmCounterKindString(uvm->counterKind),
                (unsigned long long)uvm->value,
                uvm->srcId,
                uvm->dstId);
            break;
        }
    default:
        printf("  <unknown>\n");
        break;
    }
}

static void CUPTIAPI
bufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords)
{
    uint8_t *rawBuffer;

    *size = BUF_SIZE;
    rawBuffer = (uint8_t *)malloc(*size + ALIGN_SIZE);

    *buffer = ALIGN_BUFFER(rawBuffer, ALIGN_SIZE);
    *maxNumRecords = 0;

    if (*buffer == NULL) {
        printf("Error: out of memory\n");
        exit(EXIT_FAILURE);
    }
}

static void CUPTIAPI
bufferCompleted(hipCtx_t ctx, uint32_t streamId, uint8_t *buffer, size_t size, size_t validSize)
{
    CUptiResult status;
    CUpti_Activity *record = NULL;

    do {
        status = cuptiActivityGetNextRecord(buffer, validSize, &record);
        if (status == CUPTI_SUCCESS) {
            printActivity(record);
        }
        else if (status == CUPTI_ERROR_MAX_LIMIT_REACHED) {
            break;
        }
        else {
            CUPTI_CALL(status);
        }
    } while (1);

    // report any records dropped from the queue
    size_t dropped;
    CUPTI_CALL(cuptiActivityGetNumDroppedRecords(ctx, streamId, &dropped));
    if (dropped != 0) {
        printf("Dropped %u activity records\n", (unsigned int)dropped);
    }

    free(buffer);
}

template<class T>
__host__ __device__ void checkData(const char *loc, T *data, int size, int expectedVal) {
    int i;

    for (i = 0; i < size / (int)sizeof(T); i++) {
        if (data[i] != expectedVal) {
            printf("Mismatch found on %s\n", loc);
            printf("Address 0x%p, Observed = 0x%x Expected = 0x%x\n", data+i, data[i], expectedVal);
            break;
        }
    }
}

template<class T>
__host__ __device__ void writeData(T *data, int size, int writeVal) {
    int i;

    for (i = 0; i < size / (int)sizeof(T); i++) {
        data[i] = writeVal;
    }
}

__global__ void testKernel(int *data, int size, int expectedVal)
{
    checkData("GPU", data, size, expectedVal);
    writeData(data, size, -expectedVal);
}

int main(int argc, char **argv)
{
    CUptiResult res;
    int deviceCount;
    int *data = NULL;
    int size = 64*1024;     // 64 KB
    int i = 123;
    CUpti_ActivityUnifiedMemoryCounterConfig config[2];

    DRIVER_API_CALL(hipInit(0));

    DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        exit(EXIT_WAIVED);
    }

    // register cupti activity buffer callbacks
    CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted));

    // configure unified memory counters
    config[0].scope = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_SCOPE_PROCESS_SINGLE_DEVICE;
    config[0].kind = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_HTOD;
    config[0].deviceId = 0;
    config[0].enable = 1;

    config[1].scope = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_SCOPE_PROCESS_SINGLE_DEVICE;
    config[1].kind = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_DTOH;
    config[1].deviceId = 0;
    config[1].enable = 1;

    res = cuptiActivityConfigureUnifiedMemoryCounter(config, 2);
    if (res == CUPTI_ERROR_UM_PROFILING_NOT_SUPPORTED) {
        printf("Test is waived, unified memory is not supported on the underlying platform.\n");
        exit(EXIT_WAIVED);
    }
    else if (res == CUPTI_ERROR_UM_PROFILING_NOT_SUPPORTED_ON_DEVICE) {
        printf("Test is waived, unified memory is not supported on the device.\n");
        exit(EXIT_WAIVED);
    }
    else if (res == CUPTI_ERROR_UM_PROFILING_NOT_SUPPORTED_ON_NON_P2P_DEVICES) {
        printf("Test is waived, unified memory is not supported on the non-P2P multi-gpu setup.\n");
        exit(EXIT_WAIVED);
    }
    else {
        CUPTI_CALL(res);
    }

    // enable unified memory counter activity
    CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_UNIFIED_MEMORY_COUNTER));

    // allocate unified memory
    printf("Allocation size in bytes %d\n", size);
    RUNTIME_API_CALL(hipMallocManaged(&data, size));

    // CPU access
    writeData(data, size, i);
    // kernel launch
    testKernel<<<1,1>>>(data, size, i);
    RUNTIME_API_CALL(hipDeviceSynchronize());
    // CPU access
    checkData("CPU", data, size, -i);

    // free unified memory
    RUNTIME_API_CALL(hipFree(data));

    CUPTI_CALL(cuptiActivityFlushAll(0));

    // disable unified memory counter activity
    CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_UNIFIED_MEMORY_COUNTER));

    hipDeviceReset();

    exit(EXIT_SUCCESS);
}
