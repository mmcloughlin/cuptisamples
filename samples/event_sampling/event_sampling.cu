#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2021 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain profiler
 * event values by sampling.
 */


#ifdef _WIN32
    #ifndef WIN32_LEAN_AND_MEAN
        #define WIN32_LEAN_AND_MEAN
    #endif
#endif

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <cupti_events.h>
#include <stdlib.h>

#ifdef _WIN32
#include <windows.h>
#else
#include <unistd.h>
#include <pthread.h>
#include <semaphore.h>
#endif

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

#define CHECK_CU_ERROR(err, cufunc)                                     \
  if (err != hipSuccess)                                              \
    {                                                                   \
      printf ("Error %d for CUDA Driver API function '%s'.\n",          \
              err, cufunc);                                             \
      exit(EXIT_FAILURE);                                               \
    }

#define CHECK_CUPTI_ERROR(err, cuptifunc)                       \
  if (err != CUPTI_SUCCESS)                                     \
    {                                                           \
      const char *errstr;                                       \
      cuptiGetResultString(err, &errstr);                       \
      printf ("%s:%d:Error %s for CUPTI API function '%s'.\n",  \
              __FILE__, __LINE__, errstr, cuptifunc);           \
      exit(EXIT_FAILURE);                                       \
    }

#define EVENT_NAME "inst_executed"
#define N 100000
#define ITERATIONS 10000
#define SAMPLE_PERIOD_MS 50

#ifdef _WIN32
HANDLE semaphore;
DWORD ret;
#else
sem_t semaphore;
int ret;
#endif

// used to signal from the compute thread to the sampling thread
static volatile int testComplete = 0;

static hipCtx_t context;
static hipDevice_t device;
static const char *eventName;

// Device code
__global__ void VecAdd(const int* A, const int* B, int* C, int size)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  for(int n = 0 ; n < 100; n++) {
    if (i < size)
      C[i] = A[i] + B[i];
  }
}

static void
initVec(int *vec, int n)
{
  for (int i=0; i< n; i++)
    vec[i] = i;
}

void *
sampling_func(void *arg)
{
  CUptiResult cuptiErr;
  CUpti_EventGroup eventGroup;
  CUpti_EventID eventId;
  size_t bytesRead, valueSize;
  uint32_t numInstances = 0, j = 0;
  uint64_t *eventValues = NULL, eventVal = 0;
  uint32_t profile_all = 1;

  cuptiErr = cuptiSetEventCollectionMode(context,
                                         CUPTI_EVENT_COLLECTION_MODE_CONTINUOUS);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiSetEventCollectionMode");

  cuptiErr = cuptiEventGroupCreate(context, &eventGroup, 0);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupCreate");

  cuptiErr = cuptiEventGetIdFromName(device, eventName, &eventId);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGetIdFromName");

  cuptiErr = cuptiEventGroupAddEvent(eventGroup, eventId);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupAddEvent");

  cuptiErr = cuptiEventGroupSetAttribute(eventGroup,
                                         CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
                                         sizeof(profile_all), &profile_all);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupSetAttribute");

  cuptiErr = cuptiEventGroupEnable(eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupEnable");

  valueSize = sizeof(numInstances);
  cuptiErr = cuptiEventGroupGetAttribute(eventGroup,
                                         CUPTI_EVENT_GROUP_ATTR_INSTANCE_COUNT,
                                         &valueSize, &numInstances);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupGetAttribute");

  bytesRead = sizeof(uint64_t) * numInstances;
  eventValues = (uint64_t *) malloc(bytesRead);
  if (eventValues == NULL) {
      printf("%s:%d: Failed to allocate memory.\n", __FILE__, __LINE__);
      exit(EXIT_FAILURE);
  }

  // Release the semaphore as sampling thread is ready to read events
#ifdef _WIN32
  ret = ReleaseSemaphore(semaphore, 1, NULL);
  if (ret == 0) {
    printf("Failed to release the semaphore\n");
    exit(EXIT_FAILURE);
  }
#else
  ret = sem_post(&semaphore);
  if (ret != 0) {
    printf("Failed to release the semaphore\n");
    exit(EXIT_FAILURE);
  }
#endif

  do {
    cuptiErr = cuptiEventGroupReadEvent(eventGroup,
                                        CUPTI_EVENT_READ_FLAG_NONE,
                                        eventId, &bytesRead, eventValues);
    CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupReadEvent");
    if (bytesRead != (sizeof(uint64_t) * numInstances)) {
      printf("Failed to read value for \"%s\"\n", eventName);
      exit(EXIT_FAILURE);
    }

    for (j = 0; j < numInstances; j++) {
      eventVal += eventValues[j];
    }
    printf("%s: %llu\n", eventName, (unsigned long long)eventVal);
#ifdef _WIN32
    Sleep(SAMPLE_PERIOD_MS);
#else
    usleep(SAMPLE_PERIOD_MS * 1000);
#endif
  } while (!testComplete);
  cuptiErr = cuptiEventGroupDisable(eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDisable");

  cuptiErr = cuptiEventGroupDestroy(eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDestroy");

  free(eventValues);
  return NULL;
}

static void
compute(int iters)
{
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int sum, i;
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;

  // Allocate input vectors h_A and h_B in host memory
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel (multiple times to make sure we have time for
  // sampling)
  threadsPerBlock = 256;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  for (i = 0; i < iters; i++) {
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
  }

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // Verify result
  for (i = 0; i < N; ++i) {
    sum = h_A[i] + h_B[i];
    if (h_C[i] != sum) {
      printf("kernel execution FAILED\n");
      exit(EXIT_FAILURE);
    }
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);
}

int
main(int argc, char *argv[])
{
#ifdef _WIN32
  HANDLE hThread;
#else
  int status;
  pthread_t pThread;
#endif
  hipError_t err;
  int deviceNum;
  int deviceCount;
  char deviceName[256];
  int major;
  int minor;

  printf("Usage: %s [device_num] [event_name]\n", argv[0]);

  err = hipInit(0);
  CHECK_CU_ERROR(err, "hipInit");

  err = hipGetDeviceCount(&deviceCount);
  CHECK_CU_ERROR(err, "hipGetDeviceCount");

  if (deviceCount == 0) {
    printf("There is no device supporting CUDA.\n");
    exit(EXIT_WAIVED);
  }

  if (argc > 1)
    deviceNum = atoi(argv[1]);
  else
    deviceNum = 0;
  printf("CUDA Device Number: %d\n", deviceNum);

  err = hipDeviceGet(&device, deviceNum);
  CHECK_CU_ERROR(err, "hipDeviceGet");

  err = hipDeviceGetName(deviceName, 256, device);
  CHECK_CU_ERROR(err, "hipDeviceGetName");

  printf("CUDA Device Name: %s\n", deviceName);

  err = hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
  CHECK_CU_ERROR(err, "hipDeviceGetAttribute");

  err = hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
  CHECK_CU_ERROR(err, "hipDeviceGetAttribute");

  printf("Compute Capability of Device: %d.%d\n", major,minor);
  int deviceComputeCapability = 10 * major + minor;
  if(deviceComputeCapability > 72) {
    printf("Sample unsupported on Device with compute capability > 7.2\n");
    exit(EXIT_WAIVED);
  }

  if (argc > 2) {
    eventName = argv[2];
  }
  else {
    eventName = EVENT_NAME;
  }

  err = hipCtxCreate(&context, 0, device);
  CHECK_CU_ERROR(err, "hipCtxCreate");

  // Create semaphore
#ifdef _WIN32
  semaphore = CreateSemaphore(NULL, 0, 10, NULL);
  if (semaphore == NULL) {
    printf("Failed to create the semaphore\n");
    exit(EXIT_FAILURE);
  }
#else
  ret = sem_init(&semaphore, 0, 0);
  if (ret != 0) {
    printf("Failed to create the semaphore\n");
    exit(EXIT_FAILURE);
  }
#endif

  testComplete = 0;

  printf("Creating sampling thread\n");
#ifdef _WIN32
  hThread = CreateThread(NULL, 0, (LPTHREAD_START_ROUTINE) sampling_func,
                         NULL, 0, NULL );
  if (!hThread) {
    printf("CreateThread failed\n");
    exit(EXIT_FAILURE);
  }
#else
  status = pthread_create(&pThread, NULL, sampling_func, NULL);
  if (status != 0) {
    perror("pthread_create");
    exit(EXIT_FAILURE);
  }
#endif

  // Wait for sampling thread to be ready for event collection
#ifdef _WIN32
  ret = WaitForSingleObject(semaphore, INFINITE);
  if (ret != WAIT_OBJECT_0) {
    printf("Failed to wait for the semaphore\n");
    exit(EXIT_FAILURE);
  }
#else
  ret = sem_wait(&semaphore);
  if (ret != 0) {
    printf("Failed to wait for the semaphore\n");
    exit(EXIT_FAILURE);
  }
#endif

  // run kernel while sampling
  compute(ITERATIONS);

  // "signal" the sampling thread to exit and wait for it
  testComplete = 1;
#ifdef _WIN32
  WaitForSingleObject(hThread, INFINITE);
#else
  pthread_join(pThread, NULL);
#endif

  hipDeviceSynchronize();
  exit(EXIT_SUCCESS);
}

