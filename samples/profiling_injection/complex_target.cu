#include "hip/hip_runtime.h"
// Copyright 2021 NVIDIA Corporation. All rights reserved
//
// This is a sample CUDA application with several different kernel launch
// patterns - launching on the default stream, multple streams, and multiple
// threads on different devices, if more than one device is present.
//
// The injection sample shared library can be used on this sample application,
// demonstrating that the injection code handles multple streams and multiple
// threads.

// Standard CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/driver_types.h"

// Standard STL headers
#include <chrono>
#include <cstdint>
#include <iostream>
using ::std::cerr;
using ::std::cout;
using ::std::endl;

#include <string>
using ::std::string;

#include <thread>
using ::std::thread;

#include <vector>
using ::std::vector;

#include <stdlib.h>

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

// Helpful error handlers for standard CUDA runtime calls
#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define MEMORY_ALLOCATION_CALL(var)                                             \
do {                                                                            \
    if (var == NULL) {                                                          \
        fprintf(stderr, "%s:%d: Error: Memory Allocation Failed \n",            \
                __FILE__, __LINE__);                                            \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

// Per-device configuration, buffers, stream and device information, and device pointers
typedef struct {
    int deviceID;
    hipCtx_t context;         //!< CUDA driver context, or NULL if default context has already been initialized
    vector<hipStream_t> streams;           // Each device needs its own streams
    vector<double *> d_x;                   // And device memory allocation
    vector<double *> d_y;                   // ..
} perDeviceData;

#define DAXPY_REPEAT 32768
// Loop over array of elements performing daxpy multiple times
// To be launched with only one block (artificially increasing serial time to better demonstrate overlapping replay)
__global__ void daxpyKernel(int elements, double a, double * x, double * y)
{
    for (int i = threadIdx.x; i < elements; i += blockDim.x)
        // Artificially increase kernel runtime to emphasize concurrency
        for (int j = 0; j < DAXPY_REPEAT; j++)
            y[i] = a * x[i] + y[i]; // daxpy
}

// Initialize kernel values
double a = 2.5;

// Normally you would want multiple warps, but to emphasize concurrency with streams and multiple devices
// we run the kernels on a single warp.
int threadsPerBlock = 32;
int threadBlocks = 1;

// Configurable number of kernels (streams, when running concurrently)
int const numKernels = 4;
int const numStreams = numKernels;
vector<size_t> elements(numKernels);

// Each kernel call allocates and computes (call number) * (blockSize) elements
// For 4 calls, this is 4k elements * 2 arrays * (1 + 2 + 3 + 4 stream mul) * 8B/elem =~ 640KB
int const blockSize = 4 * 1024;

// Wrapper which will launch numKernel kernel calls on a single device
// The device streams vector is used to control which stream each call is made on
// If 'serial' is non-zero, the device streams are ignored and instead the default stream is used
void launchKernels(perDeviceData &d, char const * const rangeName, bool serial)
{
    // Switch to desired device
    RUNTIME_API_CALL(hipSetDevice(d.deviceID));
    DRIVER_API_CALL(hipCtxSetCurrent(d.context));

    for (unsigned int stream = 0; stream < d.streams.size(); stream++)
    {
        hipStream_t streamId = (serial ? 0 : d.streams[stream]);
        daxpyKernel <<<threadBlocks, threadsPerBlock, 0, streamId>>> (elements[stream], a, d.d_x[stream], d.d_y[stream]);
        RUNTIME_API_CALL(hipGetLastError());
    }

    // After launching all work, synchronize all streams
    if (serial == false)
    {
        for (unsigned int stream = 0; stream < d.streams.size(); stream++)
        {
            RUNTIME_API_CALL(hipStreamSynchronize(d.streams[stream]));
        }
    }
    else
    {
        RUNTIME_API_CALL(hipStreamSynchronize(0));
    }
}


int main(int argc, char * argv[])
{
    int numDevices;
    RUNTIME_API_CALL(hipGetDeviceCount(&numDevices));

    // Per-device information
    vector<int> device_ids;

    // Find all devices
    for (int i = 0; i < numDevices; i++)
    {
        // Record device number
        device_ids.push_back(i);
    }

    numDevices = device_ids.size();
    cout << "Found " << numDevices << " devices" << endl;

    // Ensure we found at least one device
    if (numDevices == 0)
    {
        cerr << "No devices detected" << endl;
        exit(EXIT_WAIVED);
    }

    // Initialize kernel input to some known numbers
    vector<double> h_x(blockSize * numKernels);
    vector<double> h_y(blockSize * numKernels);
    for (size_t i = 0; i < blockSize * numKernels; i++)
    {
        h_x[i] = 1.5 * i;
        h_y[i] = 2.0 * (i - 3000);
    }

    // Initialize a vector of 'default stream' values to demonstrate serialized kernels
    vector<hipStream_t> defaultStreams(numStreams);
    for (int stream = 0; stream < numStreams; stream++)
    {
        defaultStreams[stream] = 0;
    }

    // Scale per-kernel work by stream number
    for (int stream = 0; stream < numStreams; stream++)
    {
        elements[stream] = blockSize * (stream + 1);
    }

    // For each device, configure profiling, set up buffers, copy kernel data
    vector<perDeviceData> deviceData(numDevices);

    for (int device = 0; device < numDevices; device++)
    {
        RUNTIME_API_CALL(hipSetDevice(device_ids[device]));
        cout << "Configuring device " << device_ids[device] << endl;

        // For simplicity's sake, in this sample, a single config struct is created per device
        deviceData[device].deviceID = device_ids[device];// GPU device ID

        DRIVER_API_CALL(hipCtxCreate(&(deviceData[device].context), 0, device_ids[device])); // Either set to a context, or may be NULL if a default context has been created

        // Per-stream initialization & memory allocation - copy from constant host array to each device array
        deviceData[device].streams.resize(numStreams);
        deviceData[device].d_x.resize(numStreams);
        deviceData[device].d_y.resize(numStreams);
        for (int stream = 0; stream < numStreams; stream++)
        {
            RUNTIME_API_CALL(hipStreamCreate(&(deviceData[device].streams[stream])));

            // Each kernel does (stream #) * blockSize work on doubles
            size_t size = elements[stream] * sizeof(double);

            RUNTIME_API_CALL(hipMalloc(&(deviceData[device].d_x[stream]), size));
            MEMORY_ALLOCATION_CALL(deviceData[device].d_x[stream]); // Validate pointer
            RUNTIME_API_CALL(hipMemcpy(deviceData[device].d_x[stream], h_x.data(), size, hipMemcpyHostToDevice));

            RUNTIME_API_CALL(hipMalloc(&(deviceData[device].d_y[stream]), size));
            MEMORY_ALLOCATION_CALL(deviceData[device].d_y[stream]); // Validate pointer
            RUNTIME_API_CALL(hipMemcpy(deviceData[device].d_y[stream], h_x.data(), size, hipMemcpyHostToDevice));
        }
    }

    //
    // First version - single device, kernel calls serialized on default stream
    //

    // Use wallclock time to measure performance
    auto begin_time = ::std::chrono::high_resolution_clock::now();

    // Run on first device and use default streams - will show runtime without any concurrency
    launchKernels(deviceData[0], "single_gpu_serial", true);

    auto end_time = ::std::chrono::high_resolution_clock::now();
    auto elapsed_serial_ms = ::std::chrono::duration_cast<::std::chrono::milliseconds>(end_time - begin_time);
    cout << "It took " << elapsed_serial_ms.count() << "ms on the host to launch " << numKernels << " kernels in serial" << endl;

    //
    // Second version - same kernel calls as before on the same device, but now using separate streams for concurrency
    // (Should be limited by the longest running kernel)
    //

    begin_time = ::std::chrono::high_resolution_clock::now();

    // Still only use first device, but this time use its allocated streams for parallelism
    launchKernels(deviceData[0], "single_gpu_async", false);

    end_time = ::std::chrono::high_resolution_clock::now();
    auto elapsed_single_device_ms = ::std::chrono::duration_cast<::std::chrono::milliseconds>(end_time - begin_time);
    cout << "It took " << elapsed_single_device_ms.count() << "ms on the host to launch " << numKernels << " kernels on a single device on separate streams" << endl;

    //
    // Third version - same as the second case, but duplicate the work across devices to show cross-device concurrency
    // This is done using threads so no serialization is needed between devices
    // (Should have roughly the same runtime as second case)
    //

    // Time creation of the same multiple streams * multiple devices
    vector<::std::thread> threads;
    begin_time = ::std::chrono::high_resolution_clock::now();

    // Now launch parallel thread work, duplicated on one thread per gpu
    for (int device = 0; device < numDevices; device++)
    {
        threads.push_back(::std::thread(launchKernels, ::std::ref(deviceData[device]), "multi_gpu_async", false));
    }

    // Wait for all threads to finish
    for (auto &t: threads)
    {
        t.join();
    }

    // Record time used when launching on multiple devices
    end_time = ::std::chrono::high_resolution_clock::now();
    auto elapsed_multiple_device_ms = ::std::chrono::duration_cast<::std::chrono::milliseconds>(end_time - begin_time);
    cout << "It took " << elapsed_multiple_device_ms.count() << "ms on the host to launch the same " << numKernels << " kernels on each of the " << numDevices << " devices in parallel" << endl;

    // Free stream memory for each device
    for (int i = 0; i < numDevices; i++)
    {
        for (int j = 0; j < numKernels; j++)
        {
            RUNTIME_API_CALL(hipFree(deviceData[i].d_x[j]));
            RUNTIME_API_CALL(hipFree(deviceData[i].d_y[j]));
        }
    }

    exit(EXIT_SUCCESS);
}
