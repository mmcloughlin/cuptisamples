#include "hip/hip_runtime.h"
/*
* Copyright 2015-2016 NVIDIA Corporation. All rights reserved.
*
* Sample to demonstrate use of NVlink CUPTI APIs
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <cupti.h>

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

#define CUPTI_CALL(call)                                                      \
do {                                                                          \
    CUptiResult _status = call;                                               \
    if (_status != CUPTI_SUCCESS) {                                           \
        const char *errstr;                                                   \
        cuptiGetResultString(_status, &errstr);                               \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",  \
                __FILE__, __LINE__, #call, errstr);                           \
        exit(EXIT_FAILURE);                                                   \
    }                                                                         \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                     \
    }                                                                          \
} while (0)

#define MEMORY_ALLOCATION_CALL(var)                                            \
do {                                                                           \
    if (var == NULL) {                                                         \
        fprintf(stderr, "%s:%d: Error: Memory Allocation Failed \n",           \
                __FILE__, __LINE__);                                           \
        exit(EXIT_FAILURE);                                                     \
    }                                                                          \
} while (0)

#define MAX_DEVICES    (32)
#define BLOCK_SIZE     (1024)
#define GRID_SIZE      (512)
#define DATA           (1024 * 1024)
#define BUF_SIZE       (32 * 1024)
#define ALIGN_SIZE     (8)
#define SUCCESS        (0)
#define NUM_METRIC     (4)
#define NUM_EVENTS     (2)
#define MAX_SIZE       (64*1024*1024)   // 64 MB
#define NUM_STREAMS    (6)   // gp100 has 6 physical copy engines
#define DATA           (1024 * 1024)

CUpti_ActivityNvLink4 *nvlinkRec = NULL;
int cpuToGpu = 0;
int gpuToGpu = 0;
int cpuToGpuAccess = 0;
int gpuToGpuAccess = 0;
bool metricSupport = true;

extern "C" __global__ void test_nvlink_bandwidth(float *src, float *dst)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = src[idx] * 2.0f;
}

static void printActivity(CUpti_Activity *record)
{
    uint32_t i = 0;
    if (record->kind == CUPTI_ACTIVITY_KIND_NVLINK) {
        nvlinkRec = (CUpti_ActivityNvLink4 *)record;

        printf("typeDev0 %d, typeDev1 %d, sysmem %d, peer %d, physical links %d, ",
          nvlinkRec->typeDev0,
          nvlinkRec->typeDev1,
          ((nvlinkRec->flag & CUPTI_LINK_FLAG_SYSMEM_ACCESS) ? 1 : 0),
          ((nvlinkRec->flag & CUPTI_LINK_FLAG_PEER_ACCESS) ? 1 : 0),
          nvlinkRec->physicalNvLinkCount);

        printf("portDev0 ");
        for (i = 0 ; i < nvlinkRec->physicalNvLinkCount ; i++ ) {
            printf("%d, ", nvlinkRec->portDev0[i]);
        }

        printf("portDev1 ");
        for (i = 0 ; i < nvlinkRec->physicalNvLinkCount ; i++ ) {
            printf("%d, ", nvlinkRec->portDev1[i]);
        }

        printf("bandwidth %llu\n", (long long unsigned int)nvlinkRec->bandwidth);

        cpuToGpuAccess |= (nvlinkRec->flag & CUPTI_LINK_FLAG_SYSMEM_ACCESS);
        gpuToGpuAccess |= (nvlinkRec->flag & CUPTI_LINK_FLAG_PEER_ACCESS);
    }
    else {
        printf("Error : Unexpected CUPTI activity kind.\nExpected Activity kind : CUPTI_ACTIVITY_KIND_NVLINK\n");
    }
}

static void CUPTIAPI bufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords)

{
    *size = BUF_SIZE + ALIGN_SIZE;
    *buffer = (uint8_t*) calloc(1, *size);
    MEMORY_ALLOCATION_CALL(*buffer);
    *maxNumRecords = 0;
}

static void CUPTIAPI bufferCompleted(hipCtx_t ctx, uint32_t streamId,
                                uint8_t *buffer, size_t size,
                                size_t validSize)
{
    CUptiResult status;
    CUpti_Activity *record = NULL;
    do {
        status = cuptiActivityGetNextRecord(buffer, validSize, &record);
        if(status == CUPTI_SUCCESS) {
            printActivity(record);
        }
        else if (status == CUPTI_ERROR_MAX_LIMIT_REACHED) {
            break;
        }
        else {
            CUPTI_CALL(status);
        }
    } while (1);

    size_t dropped;
    CUPTI_CALL(cuptiActivityGetNumDroppedRecords(ctx, streamId, &dropped));
    if (dropped != 0) {
        printf("Dropped %u activity records\n", (unsigned int)dropped);
    }
}

#define DIM(x) (sizeof(x)/sizeof(*(x)))

void calculateSize(char *result, uint64_t size)
{
    int i;

    const char *sizes[]   = { "TB", "GB", "MB", "KB", "B" };
    uint64_t  exbibytes = 1024ULL * 1024ULL * 1024ULL * 1024ULL;

    uint64_t  multiplier = exbibytes;

    for (i = 0; (unsigned)i < DIM(sizes); i++, multiplier /= (uint64_t)1024)
    {
        if (size < multiplier)
            continue;
        sprintf(result, "%.1f %s", (float) size / multiplier, sizes[i]);
        return;
    }
    strcpy(result, "0");
    return;
}

void readMetricValue(CUpti_EventGroup eventGroup, uint32_t numEvents,
                    hipDevice_t dev, CUpti_MetricID *metricId,
                    uint64_t timeDuration,
                    CUpti_MetricValue *metricValue) {

    size_t bufferSizeBytes, numCountersRead;
    uint64_t *eventValueArray = NULL;
    CUpti_EventID *eventIdArray;
    size_t arraySizeBytes = 0;
    size_t numTotalInstancesSize = 0;
    uint64_t numTotalInstances = 0;
    uint64_t *aggrEventValueArray = NULL;
    size_t aggrEventValueArraySize;
    uint32_t i = 0, j = 0;
    CUpti_EventDomainID domainId;
    size_t domainSize;

    domainSize = sizeof(CUpti_EventDomainID);

    CUPTI_CALL(cuptiEventGroupGetAttribute(eventGroup,
                                           CUPTI_EVENT_GROUP_ATTR_EVENT_DOMAIN_ID,
                                           &domainSize,
                                           (void *)&domainId));

    numTotalInstancesSize = sizeof(uint64_t);

    CUPTI_CALL(cuptiDeviceGetEventDomainAttribute(dev,
                                              domainId,
                                              CUPTI_EVENT_DOMAIN_ATTR_TOTAL_INSTANCE_COUNT,
                                              &numTotalInstancesSize,
                                              (void *)&numTotalInstances));

    arraySizeBytes = sizeof(CUpti_EventID) * numEvents;
    bufferSizeBytes = sizeof(uint64_t) * numEvents * numTotalInstances;

    eventValueArray = (uint64_t *) malloc(bufferSizeBytes);
    MEMORY_ALLOCATION_CALL(eventValueArray);

    eventIdArray = (CUpti_EventID *) malloc(arraySizeBytes);
    MEMORY_ALLOCATION_CALL(eventIdArray);

    aggrEventValueArray = (uint64_t *) calloc(numEvents, sizeof(uint64_t));
    MEMORY_ALLOCATION_CALL(aggrEventValueArray);

    aggrEventValueArraySize = sizeof(uint64_t) * numEvents;

    CUPTI_CALL(cuptiEventGroupReadAllEvents(eventGroup,
                                                CUPTI_EVENT_READ_FLAG_NONE,
                                                &bufferSizeBytes,
                                                eventValueArray,
                                                &arraySizeBytes,
                                                eventIdArray,
                                                &numCountersRead));

    for (i = 0; i < numEvents; i++) {
        for (j = 0; j < numTotalInstances; j++) {
            aggrEventValueArray[i] += eventValueArray[i + numEvents * j];
        }
    }

    for (i = 0; i < NUM_METRIC; i++) {
        CUPTI_CALL(cuptiMetricGetValue(dev, metricId[i], arraySizeBytes,
                              eventIdArray, aggrEventValueArraySize,
                              aggrEventValueArray, timeDuration,
                              &metricValue[i]));
    }

    free(eventValueArray);
    free(eventIdArray);
}

  // Print metric value, we format based on the value kind
int printMetricValue(CUpti_MetricID metricId, CUpti_MetricValue metricValue, const char *metricName) {

    CUpti_MetricValueKind valueKind;
    char str[64];
    size_t valueKindSize = sizeof(valueKind);

    CUPTI_CALL(cuptiMetricGetAttribute(metricId, CUPTI_METRIC_ATTR_VALUE_KIND,
                                       &valueKindSize, &valueKind));
    switch (valueKind) {

    case CUPTI_METRIC_VALUE_KIND_DOUBLE:
        printf("%s = ", metricName);
        calculateSize(str, (uint64_t)metricValue.metricValueDouble);
        printf("%s\n", str);
        break;

    case CUPTI_METRIC_VALUE_KIND_UINT64:
        printf("%s = ", metricName);
        calculateSize(str, (uint64_t)metricValue.metricValueUint64);
        printf("%s\n", str);
        break;

    case CUPTI_METRIC_VALUE_KIND_INT64:
        printf("%s = ", metricName);
        calculateSize(str, (uint64_t)metricValue.metricValueInt64);
        printf("%s\n", str);
        break;

    case CUPTI_METRIC_VALUE_KIND_THROUGHPUT:
        printf("%s = ", metricName);
        calculateSize(str, (uint64_t)metricValue.metricValueThroughput);
        printf("%s/Sec\n", str);
        break;

    default:
        fprintf(stderr, "error: unknown value kind\n");
        return -1;
    }
    return 0;
  }

void testCpuToGpu(CUpti_EventGroup *eventGroup, hipDeviceptr_t *pDevBuffer,
                    float** pHostBuffer, size_t bufferSize,
                    hipStream_t *cudaStreams,
                    uint64_t *timeDuration, int numEventGroup)
{
    int i;
    uint32_t value = 1;
    uint64_t startTimestamp, endTimestamp;

    for (i = 0; i < numEventGroup; i++) {
            CUPTI_CALL(cuptiEventGroupEnable(eventGroup[i]));
            CUPTI_CALL(cuptiEventGroupSetAttribute(eventGroup[i],
                                CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
                                sizeof(uint32_t), (void*)&value));
    }

    CUPTI_CALL(cuptiGetTimestamp(&startTimestamp));

    //Unidirectional copy H2D
    for (i = 0; i < NUM_STREAMS; i++)
    {
        RUNTIME_API_CALL(hipMemcpyAsync((void *)pDevBuffer[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    //Unidirectional copy D2H
    for (i = 0; i < NUM_STREAMS; i++)
    {
        RUNTIME_API_CALL(hipMemcpyAsync(pHostBuffer[i], (void *)pDevBuffer[i], bufferSize, hipMemcpyDeviceToHost, cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    //Bidirectional copy
    for (i = 0; i < NUM_STREAMS; i+=2)
    {
        RUNTIME_API_CALL(hipMemcpyAsync((void *)pDevBuffer[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
        RUNTIME_API_CALL(hipMemcpyAsync(pHostBuffer[i+1], (void *)pDevBuffer[i+1], bufferSize, hipMemcpyDeviceToHost, cudaStreams[i+1]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    CUPTI_CALL(cuptiGetTimestamp(&endTimestamp));
    *timeDuration = endTimestamp - startTimestamp;
}

void testGpuToGpu(CUpti_EventGroup *eventGroup, hipDeviceptr_t *pDevBuffer0, hipDeviceptr_t *pDevBuffer1,
                    float** pHostBuffer, size_t bufferSize,
                    hipStream_t *cudaStreams,
                    uint64_t *timeDuration, int numEventGroup)
{
    int i;
    uint32_t value = 1;
    uint64_t startTimestamp, endTimestamp;

    for (i = 0; i < numEventGroup; i++) {
        CUPTI_CALL(cuptiEventGroupEnable(eventGroup[i]));
        CUPTI_CALL(cuptiEventGroupSetAttribute(eventGroup[i],
                            CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
                            sizeof(uint32_t), (void*)&value));
    }

    RUNTIME_API_CALL(hipSetDevice(0));
    RUNTIME_API_CALL(hipDeviceEnablePeerAccess(1, 0));
    RUNTIME_API_CALL(hipSetDevice(1));
    RUNTIME_API_CALL(hipDeviceEnablePeerAccess(0, 0));

    //Unidirectional copy H2D
    for (i = 0; i < NUM_STREAMS; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *)pDevBuffer0[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    for (i = 0; i < NUM_STREAMS; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *)pDevBuffer1[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    CUPTI_CALL(cuptiGetTimestamp(&startTimestamp));

    for (i = 0; i < NUM_STREAMS; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *)pDevBuffer0[i], (void *)pDevBuffer1[i], bufferSize, hipMemcpyDeviceToDevice, cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    for (i = 0; i < NUM_STREAMS; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *)pDevBuffer1[i], (void *)pDevBuffer0[i], bufferSize, hipMemcpyDeviceToDevice, cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    for (i = 0; i < NUM_STREAMS; i++) {
        test_nvlink_bandwidth<<<GRID_SIZE, BLOCK_SIZE>>>((float*)pDevBuffer1[i], (float*)pDevBuffer0[i]);
    }

    CUPTI_CALL(cuptiGetTimestamp(&endTimestamp));
    *timeDuration = endTimestamp - startTimestamp;
}

static void printUsage() {
    printf("usage: Demonstrate use of NVlink CUPTI APIs\n");
    printf("       -help           : display help message\n");
    printf("       --cpu-to-gpu    : Show results for data transfer between CPU and GPU \n");
    printf("       --gpu-to-gpu    : Show results for data transfer between two GPUs \n");
}

void parseCommandLineArgs(int argc, char *argv[])
{
    if (argc != 2) {
        printf("Invalid number of options\n");
        exit(EXIT_FAILURE);
    }

    if (strcmp(argv[1], "--cpu-to-gpu") == 0) {
        cpuToGpu = 1;
    }
    else if (strcmp(argv[1], "--gpu-to-gpu") == 0) {
        gpuToGpu = 1;
    }
    else if ((strcmp(argv[1], "--help") == 0) ||
             (strcmp(argv[1], "-help") == 0) ||
             (strcmp(argv[1], "-h") == 0)) {
        printUsage();
        exit(EXIT_SUCCESS);
    }
    else {
        printf("Invalid/incomplete option %s\n", argv[1]);
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[])
{
    int deviceCount = 0, i = 0, j = 0, numEventGroup = 0;
    size_t bufferSize = 0, freeMemory = 0, totalMemory = 0;
    CUpti_EventGroupSets *passes = NULL;
    hipCtx_t ctx;
    char str[64];

    hipDeviceptr_t pDevBuffer0[NUM_STREAMS];
    hipDeviceptr_t pDevBuffer1[NUM_STREAMS];
    float* pHostBuffer[NUM_STREAMS];

    hipStream_t cudaStreams[NUM_STREAMS] = {0};

    CUpti_EventGroup eventGroup[32];
    CUpti_MetricID metricId[NUM_METRIC];
    uint32_t numEvents[NUM_METRIC];
    CUpti_MetricValue metricValue[NUM_METRIC];
    hipDeviceProp_t prop[MAX_DEVICES];
    uint64_t timeDuration;

    // Adding nvlink Metrics.
    const char *metricName[NUM_METRIC] = {"nvlink_total_data_transmitted",
                                    "nvlink_total_data_received",
                                    "nvlink_transmit_throughput",
                                    "nvlink_receive_throughput"};

    // Parse command line arguments
    parseCommandLineArgs(argc, argv);

    CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_NVLINK));
    CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted));

    DRIVER_API_CALL(hipInit(0));

    RUNTIME_API_CALL(hipGetDeviceCount(&deviceCount));

    printf("There are %d devices.\n", deviceCount);

    if (deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        exit(EXIT_WAIVED);
    }

    for (i = 0; i < deviceCount; i++) {
        RUNTIME_API_CALL(hipGetDeviceProperties(&prop[i], i));
        printf("CUDA Device %d Name: %s\n", i, prop[i].name);
        // Check if any device is Turing+
        if (prop[i].major == 7 && prop[i].minor > 0) {
            metricSupport = false;
        } else if (prop[i].major > 7) {
            metricSupport = false;
        }
    }

    // Set memcpy size based on available device memory
    RUNTIME_API_CALL(hipMemGetInfo(&freeMemory, &totalMemory));
    bufferSize = MAX_SIZE < (freeMemory/4) ? MAX_SIZE : (freeMemory/4);

    printf("Total Device Memory available : ");
    calculateSize(str, (uint64_t)totalMemory);
    printf("%s\n", str);

    printf("Memcpy size is set to %llu B (%llu MB)\n",
    (unsigned long long)bufferSize, (unsigned long long)bufferSize/(1024*1024));

    for(i = 0; i < NUM_STREAMS; i++) {
       RUNTIME_API_CALL(hipStreamCreate(&cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    // Nvlink-topology Records are generated even before hipMemcpy API is called.
    CUPTI_CALL(cuptiActivityFlushAll(0));

    // Transfer Data between Host And Device, if Nvlink is Present
    // Check condition : nvlinkRec->flag & CUPTI_LINK_FLAG_SYSMEM_ACCESS
    // True : Nvlink is present between CPU & GPU
    // False : Nvlink is not present.
    if ((nvlinkRec) && (((cpuToGpu) && (cpuToGpuAccess)) || ((gpuToGpu) && (gpuToGpuAccess)))) {
        if (!metricSupport) {
            printf("Legacy CUPTI metrics not supported from Turing+ devices.\n");
            exit(EXIT_WAIVED);
        }

        for (i = 0; i < NUM_METRIC; i++) {
            CUPTI_CALL(cuptiMetricGetIdFromName(0, metricName[i], &metricId[i]));
            CUPTI_CALL(cuptiMetricGetNumEvents(metricId[i], &numEvents[i]));
        }

        DRIVER_API_CALL(hipCtxCreate(&ctx, 0, 0));

        CUPTI_CALL(cuptiMetricCreateEventGroupSets(ctx, (sizeof metricId) ,metricId, &passes));

        // EventGroups required to profile Nvlink metrics.
        for (i = 0; i < (signed)passes->numSets; i++) {
            for (j = 0; j < (signed)passes->sets[i].numEventGroups; j++) {
                eventGroup[numEventGroup] = passes->sets[i].eventGroups[j];

                if (!eventGroup[numEventGroup]) {
                    printf("\n eventGroup initialization failed \n");
                    exit(EXIT_FAILURE);
                }

                numEventGroup++;
            }
        }

        CUPTI_CALL(cuptiSetEventCollectionMode(ctx, CUPTI_EVENT_COLLECTION_MODE_CONTINUOUS));

        // ===== Allocate Memory =====================================

        for(i = 0; i < NUM_STREAMS; i++) {
            RUNTIME_API_CALL(hipMalloc((void**)&pDevBuffer0[i], bufferSize));

            pHostBuffer[i] = (float *)malloc(bufferSize);
            MEMORY_ALLOCATION_CALL(pHostBuffer[i]);
        }

        if (cpuToGpu) {
            testCpuToGpu(eventGroup, pDevBuffer0, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            printf("Data tranferred between CPU & Device%d : \n", (int)nvlinkRec->typeDev0);
        }
        else if(gpuToGpu) {
            RUNTIME_API_CALL(hipSetDevice(1));
            for(i = 0; i < NUM_STREAMS; i++) {
                RUNTIME_API_CALL(hipMalloc((void**)&pDevBuffer1[i], bufferSize));
            }
            testGpuToGpu(eventGroup, pDevBuffer0, pDevBuffer1,pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            printf("Data tranferred between Device 0 & Device 1 : \n");
        }

        // Collect Nvlink Metric values for the data transfer via Nvlink for all the eventGroups.
        for (i = 0; i < numEventGroup; i++) {
            readMetricValue(eventGroup[i], NUM_EVENTS, 0, metricId, timeDuration, metricValue);

            CUPTI_CALL(cuptiEventGroupDisable(eventGroup[i]));
            CUPTI_CALL(cuptiEventGroupDestroy(eventGroup[i]));

            for (i = 0; i < NUM_METRIC; i++) {
                if (printMetricValue(metricId[i], metricValue[i], metricName[i]) != 0) {
                    printf("\n printMetricValue failed \n");
                    exit(EXIT_FAILURE);
                }
            }
        }
    }
    else {
        printf("No Nvlink supported device found\n");
        exit(EXIT_WAIVED);
    }

    exit(EXIT_SUCCESS);
}
