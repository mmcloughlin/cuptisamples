#include "hip/hip_runtime.h"
/*
* Copyright 2014-2017 NVIDIA Corporation. All rights reserved.
*
* Sample app to demonstrate use of CUPTI library to obtain profiler
* event values on a multi-gpu setup without serializing the compute
* work on the devices.
*/

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <cupti_events.h>
#include <stdlib.h>

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

#define CUPTI_CALL(call)                                                    \
do {                                                                        \
    CUptiResult _status = call;                                             \
    if (_status != CUPTI_SUCCESS) {                                         \
      const char *errstr;                                                   \
      cuptiGetResultString(_status, &errstr);                               \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",  \
              __FILE__, __LINE__, #call, errstr);                           \
      exit(EXIT_FAILURE);                                                   \
    }                                                                       \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define MAX_DEVICES     32
#define BLOCK_X         32
#define GRID_X          32

// default event
#define EVENT_NAME      "inst_executed"

// dummy kernel
__global__ void kernel() {
    uint64_t i = 0;
    volatile uint64_t limit = 1024 * 128;
    for (i = 0; i < limit; i++) {
    }
}

int
main(int argc, char *argv[])
{
    int deviceCount;
    char deviceName[256];
    hipDevice_t device[MAX_DEVICES];
    hipCtx_t context[MAX_DEVICES];
    CUpti_EventGroup eventGroup[MAX_DEVICES];
    CUpti_EventID eventId[MAX_DEVICES];
    size_t bytesRead, valueSize;
    uint32_t numInstances = 0, j = 0;
    uint64_t *eventValues = NULL, eventVal = 0;
    const char *eventName;
    int i = 0;
    uint32_t profile_all = 1;

    printf("Usage: %s [event_name]\n", argv[0]);

    DRIVER_API_CALL(hipInit(0));

    DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        exit(EXIT_WAIVED);
    }

    if (deviceCount < 2) {
        printf("This multi-gpu test is waived on single gpu setup.\n");
        exit(EXIT_WAIVED);
    }

    if (deviceCount > MAX_DEVICES) {
        printf("Found more devices (%d) than handled in the test (%d)\n",
            deviceCount, MAX_DEVICES);
        exit(EXIT_WAIVED);
    }

    if (argc > 1) {
        eventName = argv[1];
    }
    else {
        eventName = EVENT_NAME;
    }

    for (i = 0; i < deviceCount; i++) {
        DRIVER_API_CALL(hipDeviceGet(&device[i], i));

        DRIVER_API_CALL(hipDeviceGetName(deviceName, 256, device[i]));

        printf("CUDA Device Name: %s\n", deviceName);
    }

    // create one context per device
    for (i = 0; i < deviceCount; i++) {
        RUNTIME_API_CALL(hipSetDevice(i));

        DRIVER_API_CALL(hipCtxCreate(&(context[i]), 0, device[i]));

        DRIVER_API_CALL(hipCtxPopCurrent(&(context[i])));
    }

    // enable event profiling on each device
    for (i = 0; i < deviceCount; i++) {
        RUNTIME_API_CALL(hipSetDevice(i));

        DRIVER_API_CALL(hipCtxPushCurrent(context[i]));

        CUPTI_CALL(cuptiSetEventCollectionMode(context[i],
            CUPTI_EVENT_COLLECTION_MODE_KERNEL));
        CUPTI_CALL(cuptiEventGroupCreate(context[i], &eventGroup[i], 0));
        CUPTI_CALL(cuptiEventGetIdFromName(device[i], eventName, &eventId[i]));
        CUPTI_CALL(cuptiEventGroupAddEvent(eventGroup[i], eventId[i]));
        CUPTI_CALL(cuptiEventGroupSetAttribute(eventGroup[i],
                                               CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
                                               sizeof(profile_all), &profile_all));
        CUPTI_CALL(cuptiEventGroupEnable(eventGroup[i]));

        DRIVER_API_CALL(hipCtxPopCurrent(&context[i]));
    }

    // launch kernel on each device
    for (i = 0; i < deviceCount; i++) {
        RUNTIME_API_CALL(hipSetDevice(i));

        DRIVER_API_CALL(hipCtxPushCurrent(context[i]));

        kernel<<<GRID_X, BLOCK_X>>>();

        // don't do any sync here, it's done once
        // work is queued on all devices

        DRIVER_API_CALL(hipCtxPopCurrent(&context[i]));
    }

    // sync each context now
    for (i = 0; i < deviceCount; i++) {
        RUNTIME_API_CALL(hipSetDevice(i));

        DRIVER_API_CALL(hipCtxPushCurrent(context[i]));

        DRIVER_API_CALL(hipCtxSynchronize());

        DRIVER_API_CALL(hipCtxPopCurrent(&context[i]));
    }

    // read events
    for (i = 0; i < deviceCount; i++) {
        RUNTIME_API_CALL(hipSetDevice(i));

        DRIVER_API_CALL(hipCtxPushCurrent(context[i]));

        valueSize = sizeof(numInstances);
        CUPTI_CALL(cuptiEventGroupGetAttribute(eventGroup[i],
                                               CUPTI_EVENT_GROUP_ATTR_INSTANCE_COUNT,
                                               &valueSize, &numInstances));

        bytesRead = sizeof(uint64_t) * numInstances;
        eventValues = (uint64_t *) malloc(bytesRead);
        if (eventValues == NULL) {
            printf("%s:%d: Failed to allocate memory.\n", __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        CUPTI_CALL(cuptiEventGroupReadEvent(eventGroup[i],
            CUPTI_EVENT_READ_FLAG_NONE,
            eventId[i], &bytesRead, eventValues));

        if (bytesRead != (sizeof(uint64_t) * numInstances)) {
            printf("Failed to read value for \"%s\"\n", eventName);
            exit(EXIT_FAILURE);
        }

        for (j = 0; j < numInstances; j++) {
            eventVal += eventValues[j];
        }

        printf("[%d] %s: %llu\n", i, eventName, (unsigned long long)eventVal);

        CUPTI_CALL(cuptiEventGroupDisable(eventGroup[i]));
        CUPTI_CALL(cuptiEventGroupDestroy(eventGroup[i]));

        DRIVER_API_CALL(hipCtxPopCurrent(&context[i]));
    }

    exit(EXIT_SUCCESS);
}
