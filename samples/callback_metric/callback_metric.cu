#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2017 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain metric values
 * using callbacks for CUDA runtime APIs
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <stdlib.h>

#define METRIC_NAME "ipc"

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                     \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define CUPTI_CALL(call)                                                \
  do {                                                                  \
    CUptiResult _status = call;                                         \
    if (_status != CUPTI_SUCCESS) {                                     \
      const char *errstr;                                               \
      cuptiGetResultString(_status, &errstr);                           \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
              __FILE__, __LINE__, #call, errstr);                       \
      exit(EXIT_FAILURE);                                               \
    }                                                                   \
  } while (0)

#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \
  (((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer))

// User data for event collection callback
typedef struct MetricData_st {
  // the device where metric is being collected
  hipDevice_t device;
  // the set of event groups to collect for a pass
  CUpti_EventGroupSet *eventGroups;
  // the current number of events collected in eventIdArray and
  // eventValueArray
  uint32_t eventIdx;
  // the number of entries in eventIdArray and eventValueArray
  uint32_t numEvents;
  // array of event ids
  CUpti_EventID *eventIdArray;
  // array of event values
  uint64_t *eventValueArray;
} MetricData_t;

static uint64_t kernelDuration;

// Device code
__global__ void VecAdd(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

static void
initVec(int *vec, int n)
{
  for (int i=0; i< n; i++)
    vec[i] = i;
}

void CUPTIAPI
getMetricValueCallback(void *userdata, CUpti_CallbackDomain domain,
                       CUpti_CallbackId cbid, const CUpti_CallbackData *cbInfo)
{
  MetricData_t *metricData = (MetricData_t*)userdata;
  unsigned int i, j, k;

  // This callback is enabled only for launch so we shouldn't see
  // anything else.
  if ((cbid != CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020) &&
      (cbid != CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000))
  {
    printf("%s:%d: unexpected cbid %d\n", __FILE__, __LINE__, cbid);
    exit(EXIT_FAILURE);
  }

  // on entry, enable all the event groups being collected this pass,
  // for metrics we collect for all instances of the event
  if (cbInfo->callbackSite == CUPTI_API_ENTER) {
    hipDeviceSynchronize();

    CUPTI_CALL(cuptiSetEventCollectionMode(cbInfo->context,
                                           CUPTI_EVENT_COLLECTION_MODE_KERNEL));

    for (i = 0; i < metricData->eventGroups->numEventGroups; i++) {
      uint32_t all = 1;
      CUPTI_CALL(cuptiEventGroupSetAttribute(metricData->eventGroups->eventGroups[i],
                                             CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
                                             sizeof(all), &all));
      CUPTI_CALL(cuptiEventGroupEnable(metricData->eventGroups->eventGroups[i]));
    }
  }

  // on exit, read and record event values
  if (cbInfo->callbackSite == CUPTI_API_EXIT) {
    hipDeviceSynchronize();

    // for each group, read the event values from the group and record
    // in metricData
    for (i = 0; i < metricData->eventGroups->numEventGroups; i++) {
      CUpti_EventGroup group = metricData->eventGroups->eventGroups[i];
      CUpti_EventDomainID groupDomain;
      uint32_t numEvents, numInstances, numTotalInstances;
      CUpti_EventID *eventIds;
      size_t groupDomainSize = sizeof(groupDomain);
      size_t numEventsSize = sizeof(numEvents);
      size_t numInstancesSize = sizeof(numInstances);
      size_t numTotalInstancesSize = sizeof(numTotalInstances);
      uint64_t *values, normalized, *sum;
      size_t valuesSize, eventIdsSize;
      size_t numCountersRead = 0;

      CUPTI_CALL(cuptiEventGroupGetAttribute(group,
                                             CUPTI_EVENT_GROUP_ATTR_EVENT_DOMAIN_ID,
                                             &groupDomainSize, &groupDomain));
      CUPTI_CALL(cuptiDeviceGetEventDomainAttribute(metricData->device, groupDomain,
                                                    CUPTI_EVENT_DOMAIN_ATTR_TOTAL_INSTANCE_COUNT,
                                                    &numTotalInstancesSize, &numTotalInstances));
      CUPTI_CALL(cuptiEventGroupGetAttribute(group,
                                             CUPTI_EVENT_GROUP_ATTR_INSTANCE_COUNT,
                                             &numInstancesSize, &numInstances));
      CUPTI_CALL(cuptiEventGroupGetAttribute(group,
                                             CUPTI_EVENT_GROUP_ATTR_NUM_EVENTS,
                                             &numEventsSize, &numEvents));
      eventIdsSize = numEvents * sizeof(CUpti_EventID);
      eventIds = (CUpti_EventID *)malloc(eventIdsSize);
      CUPTI_CALL(cuptiEventGroupGetAttribute(group,
                                             CUPTI_EVENT_GROUP_ATTR_EVENTS,
                                             &eventIdsSize, eventIds));

      valuesSize = sizeof(uint64_t) * numInstances * numEvents;
      values = (uint64_t *)malloc(valuesSize);

      CUPTI_CALL(cuptiEventGroupReadAllEvents(group,
                                          CUPTI_EVENT_READ_FLAG_NONE,
                                          &valuesSize,
                                          values,
                                          &eventIdsSize,
                                          eventIds,
                                          &numCountersRead));

      if (metricData->eventIdx >= metricData->numEvents) {
        fprintf(stderr, "error: too many events collected, metric expects only %d\n",
                (int)metricData->numEvents);
        exit(EXIT_FAILURE);
      }

      sum = (uint64_t *)calloc(sizeof(uint64_t), numEvents);
      // sum collect event values from all instances
      for (k = 0; k < numInstances; k++) {
        for (j = 0; j < numEvents; j++) {
            sum[j] += values[(k * numEvents) + j];
        }
      }

      for (j = 0; j < numEvents; j++) {
        // normalize the event value to represent the total number of
        // domain instances on the device
        normalized = (sum[j] * numTotalInstances) / numInstances;

        metricData->eventIdArray[metricData->eventIdx] = eventIds[j];
        metricData->eventValueArray[metricData->eventIdx] = normalized;
        metricData->eventIdx++;

        // print collected value
        {
          char eventName[128];
          size_t eventNameSize = sizeof(eventName) - 1;
          CUPTI_CALL(cuptiEventGetAttribute(eventIds[j], CUPTI_EVENT_ATTR_NAME,
                                            &eventNameSize, eventName));
          eventName[127] = '\0';
          printf("\t%s = %llu (", eventName, (unsigned long long)sum[j]);
          if (numInstances > 1) {
            for (k = 0; k < numInstances; k++) {
              if (k != 0)
                printf(", ");
              printf("%llu", (unsigned long long)values[(k * numEvents) + j]);
            }
          }

          printf(")\n");
          printf("\t%s (normalized) (%llu * %u) / %u = %llu\n",
                 eventName, (unsigned long long)sum[j],
                 numTotalInstances, numInstances,
                 (unsigned long long)normalized);
        }
      }

      free(values);
      free(sum);
    }

    for (i = 0; i < metricData->eventGroups->numEventGroups; i++)
      CUPTI_CALL(cuptiEventGroupDisable(metricData->eventGroups->eventGroups[i]));
  }
}

static void
cleanUp(int *h_A, int *h_B, int *h_C, int *d_A, int *d_B, int *d_C)
{
  if (d_A)
    hipFree(d_A);
  if (d_B)
    hipFree(d_B);
  if (d_C)
    hipFree(d_C);

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_B)
    free(h_B);
  if (h_C)
    free(h_C);
}

static void
runPass()
{
  int N = 50000;
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;
  int i, sum;

  // Allocate input vectors h_A and h_B in host memory
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel
  threadsPerBlock = 256;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  printf("Launching kernel: blocks %d, thread/block %d\n",
         blocksPerGrid, threadsPerBlock);

  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // Verify result
  for (i = 0; i < N; ++i) {
    sum = h_A[i] + h_B[i];
    if (h_C[i] != sum) {
      fprintf(stderr, "error: result verification failed\n");
      exit(EXIT_FAILURE);
    }
  }

  cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
}

static void CUPTIAPI
bufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords)
{
  uint8_t *rawBuffer;

  *size = 16 * 1024;
  rawBuffer = (uint8_t *)malloc(*size + ALIGN_SIZE);

  *buffer = ALIGN_BUFFER(rawBuffer, ALIGN_SIZE);
  *maxNumRecords = 0;

  if (*buffer == NULL) {
    printf("Error: out of memory\n");
    exit(EXIT_FAILURE);
  }
}

static void CUPTIAPI
bufferCompleted(hipCtx_t ctx, uint32_t streamId, uint8_t *buffer, size_t size, size_t validSize)
{
  CUpti_Activity *record = NULL;
  CUpti_ActivityKernel8 *kernel;

  //since we launched only 1 kernel, we should have only 1 kernel record
  CUPTI_CALL(cuptiActivityGetNextRecord(buffer, validSize, &record));

  kernel = (CUpti_ActivityKernel8 *)record;
  if (kernel->kind != CUPTI_ACTIVITY_KIND_KERNEL) {
    fprintf(stderr, "Error: expected kernel activity record, got %d\n", (int)kernel->kind);
    exit(EXIT_FAILURE);
  }

  kernelDuration = kernel->end - kernel->start;
  free(buffer);
}

int
main(int argc, char *argv[])
{
  CUpti_SubscriberHandle subscriber;
  hipCtx_t context = 0;
  hipDevice_t device = 0;
  int deviceNum;
  int deviceCount;
  char deviceName[256];
  const char *metricName;
  CUpti_MetricID metricId;
  CUpti_EventGroupSets *passData;
  MetricData_t metricData;
  unsigned int pass;
  CUpti_MetricValue metricValue;

  printf("Usage: %s [device_num] [metric_name]\n", argv[0]);

  // make sure activity is enabled before any CUDA API
  CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_KERNEL));

  DRIVER_API_CALL(hipInit(0));
  DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));
  if (deviceCount == 0) {
    printf("There is no device supporting CUDA.\n");
    exit(EXIT_WAIVED);
  }

  if (argc > 1)
    deviceNum = atoi(argv[1]);
  else
    deviceNum = 0;
  printf("CUDA Device Number: %d\n", deviceNum);

  DRIVER_API_CALL(hipDeviceGet(&device, deviceNum));
  DRIVER_API_CALL(hipDeviceGetName(deviceName, 256, device));
  printf("CUDA Device Name: %s\n", deviceName);

  int major, minor;
  DRIVER_API_CALL(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device));
  DRIVER_API_CALL(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device));

  int deviceComputeCapability = 10 * major + minor;
  printf("Compute Capability of Device: %d.%d\n", major,minor);
  if(deviceComputeCapability > 72) {
    printf("Sample unsupported on Device with compute capability > 7.2\n");
    exit(EXIT_WAIVED);
  }

  DRIVER_API_CALL(hipCtxCreate(&context, 0, device));

  // Get the name of the metric to collect
  if (argc > 2)
    metricName = argv[2];
  else {
    metricName = METRIC_NAME;
  }

  // need to collect duration of kernel execution without any event
  // collection enabled (some metrics need kernel duration as part of
  // calculation). The only accurate way to do this is by using the
  // activity API.
  {
    CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted));
    runPass();
    hipDeviceSynchronize();
    CUPTI_CALL(cuptiActivityFlushAll(0));
  }

  // setup launch callback for event collection
  CUPTI_CALL(cuptiSubscribe(&subscriber, (CUpti_CallbackFunc)getMetricValueCallback, &metricData));
  CUPTI_CALL(cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_RUNTIME_API,
                                 CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020));
  CUPTI_CALL(cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_RUNTIME_API,
                                 CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000));

  // allocate space to hold all the events needed for the metric
  CUPTI_CALL(cuptiMetricGetIdFromName(device, metricName, &metricId));
  CUPTI_CALL(cuptiMetricGetNumEvents(metricId, &metricData.numEvents));
  metricData.device = device;
  metricData.eventIdArray = (CUpti_EventID *)malloc(metricData.numEvents * sizeof(CUpti_EventID));
  metricData.eventValueArray = (uint64_t *)malloc(metricData.numEvents * sizeof(uint64_t));
  metricData.eventIdx = 0;

  // get the number of passes required to collect all the events
  // needed for the metric and the event groups for each pass
  CUPTI_CALL(cuptiMetricCreateEventGroupSets(context, sizeof(metricId), &metricId, &passData));
  for (pass = 0; pass < passData->numSets; pass++) {
    printf("Pass %u\n", pass);
    metricData.eventGroups = passData->sets + pass;
    runPass();
  }

  if (metricData.eventIdx != metricData.numEvents) {
    fprintf(stderr, "error: expected %u metric events, got %u\n",
            metricData.numEvents, metricData.eventIdx);
    exit(EXIT_FAILURE);
  }

  // use all the collected events to calculate the metric value
  CUPTI_CALL(cuptiMetricGetValue(device, metricId,
                                 metricData.numEvents * sizeof(CUpti_EventID),
                                 metricData.eventIdArray,
                                 metricData.numEvents * sizeof(uint64_t),
                                 metricData.eventValueArray,
                                 kernelDuration, &metricValue));

  // print metric value, we format based on the value kind
  {
    CUpti_MetricValueKind valueKind;
    size_t valueKindSize = sizeof(valueKind);
    CUPTI_CALL(cuptiMetricGetAttribute(metricId, CUPTI_METRIC_ATTR_VALUE_KIND,
                                       &valueKindSize, &valueKind));
    switch (valueKind) {
    case CUPTI_METRIC_VALUE_KIND_DOUBLE:
      printf("Metric %s = %f\n", metricName, metricValue.metricValueDouble);
      break;
    case CUPTI_METRIC_VALUE_KIND_UINT64:
      printf("Metric %s = %llu\n", metricName,
             (unsigned long long)metricValue.metricValueUint64);
      break;
    case CUPTI_METRIC_VALUE_KIND_INT64:
      printf("Metric %s = %lld\n", metricName,
             (long long)metricValue.metricValueInt64);
      break;
    case CUPTI_METRIC_VALUE_KIND_PERCENT:
      printf("Metric %s = %f%%\n", metricName, metricValue.metricValuePercent);
      break;
    case CUPTI_METRIC_VALUE_KIND_THROUGHPUT:
      printf("Metric %s = %llu bytes/sec\n", metricName,
             (unsigned long long)metricValue.metricValueThroughput);
      break;
    case CUPTI_METRIC_VALUE_KIND_UTILIZATION_LEVEL:
      printf("Metric %s = utilization level %u\n", metricName,
             (unsigned int)metricValue.metricValueUtilizationLevel);
      break;
    default:
      fprintf(stderr, "error: unknown value kind\n");
      exit(EXIT_FAILURE);
    }
  }

  CUPTI_CALL(cuptiUnsubscribe(subscriber));
  exit(EXIT_SUCCESS);
}
